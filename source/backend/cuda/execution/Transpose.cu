#include "hip/hip_runtime.h"
//
//  Transpose.cu
//  MNN
//
//  Created by MNN on b'2021/12/09'.
//  Copyright © 2018, Alibaba Group Holding Limited
//

#include "Transpose.cuh"
#include "core/Macro.h"
#include "MNNCUDADefine.hpp"
#include "MNNCUDAFunction.cuh"
namespace MNN {
namespace CUDA {

template<typename T0, typename T1>
__global__ void UNPACKCOMMON_4(const T0 *input, T1 *output,
    const int total, int inside, int axis, int outside,
    int insideStride, int axisStride, int axisAlign,
    DivModFast is, DivModFast cs
    ) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total; i += blockDim.x * gridDim.x) {
        int tmp, x, y, z;
        cs.divmod(i, tmp, y);
        is.divmod(tmp, z, x);
        if (y < axis) {
            int srcOffset = (z * inside + x) * axisAlign + y;// NHWC8 , inside <-> HW, ouside <-> N
            int dstOffset = x * insideStride + y * axisStride + z * inside * axis;
            output[dstOffset] = input[srcOffset];
        }
    }
}

template<typename T0, typename T1>
__global__ void UNPACKCOMMON(const T0 *input, T1 *output, 
    int inside, int axis, int outside, 
    int insideStride, int axisStride
    ) {
    int axisAlign = UP_DIV(axis, PACK_NUMBER) * PACK_NUMBER;;
    int total = axisAlign * inside * outside;
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total; i += blockDim.x * gridDim.x) {
        int tmpI = i / axisAlign;
        int y = i % axisAlign;
        int x = tmpI % inside;
        int z = tmpI / inside;

        int srcOffset = (z * inside + x) * axisAlign + y;// NHWC8 , inside <-> HW, ouside <-> N
        int dstOffset = x * insideStride + y * axisStride + z * inside * axis;
        if (y < axis) {
            output[dstOffset] = input[srcOffset];
        }
    }
}

__global__ void UNPACKCOMMON_REARRANGE_half_4(const double *input, double *output,
    int inside, int axis, int outside,
    int insideStride, int axisStride
    ) {
    int axisAlign = UP_DIV(axis, PACK_NUMBER) * PACK_NUMBER / 4;
    int insideAlign = inside / 4;
    int axisNum = axis / 16;
    int insideNum = inside / 32;

    __shared__ double sharedData[128];
    int tid = blockIdx.x;
    int localIdx = threadIdx.x;

    int tmpI = tid / axisNum;
    int y = tid % axisNum;
    int x = tmpI % insideNum;
    int z = tmpI / insideNum;

    int y_mod = localIdx % 4; // [0 ~ 3]
    int x_mod = localIdx / 4; // [0 ~ 31]
    int srcOffset = (z * inside + 32*x+x_mod) * axisAlign + 4*y+y_mod;
    sharedData[localIdx] = input[srcOffset];// [HW_32, C4_4]

    __syncthreads();

    int oy_mod = localIdx % 16; // [0 ~ 15]
    int ox_mod = localIdx / 16; // [0 ~ 7]

    // [4*ox_mod, oy_mod]
    half tmp_data[4];
    tmp_data[0] = ((half*)sharedData)[(4*ox_mod+0) * 16 + oy_mod];
    tmp_data[1] = ((half*)sharedData)[(4*ox_mod+1) * 16 + oy_mod];
    tmp_data[2] = ((half*)sharedData)[(4*ox_mod+2) * 16 + oy_mod];
    tmp_data[3] = ((half*)sharedData)[(4*ox_mod+3) * 16 + oy_mod];

    int dstOffset = (8*x+ox_mod) + (z * axis + (16*y+oy_mod)) * insideAlign;

    output[dstOffset] = ((double*)tmp_data)[0];
}

template<typename T0, typename T1>
__global__ void PACKCOMMON_4(const T0 *input, T1 *output, 
    int inside, int axis, int outside, 
    int insideStride, int axisStride,
    DivModFast is, DivModFast cs
    ) {
    int axisAlign = UP_DIV(axis, PACK_NUMBER/ 4) * PACK_NUMBER / 4;;
    int total = axisAlign * inside * outside;

    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total; i += blockDim.x * gridDim.x) {
        int tmp, x, y, z;
        cs.divmod(i, tmp, y);
        is.divmod(tmp, z, x);
        int dstOffset = (z * inside + x) * axisAlign + y;
        int srcOffset = x * insideStride + y * axisStride + z * inside * axis;
        if (y < axis) {
            output[dstOffset] = input[srcOffset];
        } else {
            output[dstOffset] = {0, 0, 0, 0};
        }
    }
}

template<typename T0, typename T1>
__global__ void PACKCOMMON_half_4(const T0 *input, T1 *output,
    int inside, int axis, int outside,
    int insideStride, int axisStride,
    DivModFast is, DivModFast cs
    ) {
    int axisAlign = UP_DIV(axis, PACK_NUMBER/ 4) * PACK_NUMBER / 4;;
    int total = axisAlign * inside * outside;

    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total; i += blockDim.x * gridDim.x) {
        int tmp, x, y, z;
        cs.divmod(i, tmp, y);
        is.divmod(tmp, z, x);
        int dstOffset = (z * inside + x) * axisAlign + y;
        int srcOffset = x * insideStride + y * axisStride + z * inside * axis;
        if (y < axis) {
            output[dstOffset] = input[srcOffset];
        } else {
            output[dstOffset] = {0, 0};
        }
    }
}

template<typename T0, typename T1>
__global__ void PACKCOMMON(const T0 *input, T1 *output,
    int inside, int axis, int outside, 
    int insideStride, int axisStride
    ) {
    int axisAlign = UP_DIV(axis, PACK_NUMBER) * PACK_NUMBER;;
    int total = axisAlign * inside * outside;
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total; i += blockDim.x * gridDim.x) {
        int tmpI = i / axisAlign;
        int y = i % axisAlign;
        int x = tmpI % inside;
        int z = tmpI / inside;

        int dstOffset = (z * inside + x) * axisAlign + y;
        int srcOffset = x * insideStride + y * axisStride + z * inside * axis;
        if (y < axis) {
            output[dstOffset] = input[srcOffset];
        } else {
            output[dstOffset] = 0.0;
        }
    }
}

__global__ void PACKCOMMON_REARRANGE_half_4(const double *input, double *output,
    int inside, int axis, int outside,
    int insideStride, int axisStride
    ) {
    int axisAlign = UP_DIV(axis, PACK_NUMBER) * PACK_NUMBER / 4;

    int insideAlign = inside / 4;
    int axisNum = axis / 16;
    int insideNum = inside / 32;

    __shared__ double sharedData[128];
    int tid = blockIdx.x;
    int localIdx = threadIdx.x;

    int tmpI = tid / axisNum;
    int y = tid % axisNum;
    int x = tmpI % insideNum;
    int z = tmpI / insideNum;

    int x_mod = localIdx % 8; // [0 ~ 8]
    int y_mod = localIdx / 8; // [0 ~ 15]
    int srcOffset = (8*x+x_mod) + (z * axis + (16*y+y_mod)) * insideAlign;
    sharedData[localIdx] = input[srcOffset];// [C_16, HW4_8]

    __syncthreads();

    int oy_mod = localIdx % 4; // [0 ~ 3]
    int ox_mod = localIdx / 4; // [0 ~ 31]
    int dstOffset = (z * inside + 32*x+ox_mod) * axisAlign + 4*y+oy_mod;

    // [4*oy_mod, ox_mod]
    half tmp_data[4];
    tmp_data[0] = ((half*)sharedData)[(4*oy_mod+0) * 32 + ox_mod];
    tmp_data[1] = ((half*)sharedData)[(4*oy_mod+1) * 32 + ox_mod];
    tmp_data[2] = ((half*)sharedData)[(4*oy_mod+2) * 32 + ox_mod];
    tmp_data[3] = ((half*)sharedData)[(4*oy_mod+3) * 32 + ox_mod];

    output[dstOffset] = ((double*)tmp_data)[0];
}

void PackBuffer(void* output, const void* input, const PackInfo* info, int bytes, CUDARuntime* runtime) {
    auto& prop = runtime->prop();
    int cores = prop.multiProcessorCount;
    int threadNumbers = prop.maxThreadsPerBlock;
    
    if (info->axis % 4 == 0 && info->axisStride == 1 && \
        info->insideStride == info->axis) {
        
        int axis_pack = UP_DIV(info->axis, PACK_NUMBER) * PACK_NUMBER / 4;
        DivModFast is(info->inside);
        DivModFast cs(axis_pack);
	if(bytes == 4) {
            PACKCOMMON_4<<<cores, threadNumbers>>>((const int4*)input, (int4*)output,
                    info->inside, info->axis / 4, info->outside, 
                    info->insideStride / 4, info->axisStride, is, cs);
            checkKernelErrors;
	    return;
	}
        if(bytes == 2) {
            PACKCOMMON_half_4<<<cores, threadNumbers>>>((const int2*)input, (int2*)output,
                    info->inside, info->axis / 4, info->outside,
                    info->insideStride / 4, info->axisStride, is, cs);
            checkKernelErrors;
            return;
        }
    }
    if (info->axis % 16 == 0 && info->inside % 32 == 0 && info->insideStride == 1 && info->axisStride == info->inside && bytes == 2) {
        int thread=128;
        int block=info->axis/16 * info->inside/32 * info->outside;
        PACKCOMMON_REARRANGE_half_4<<<block, thread>>>((const double*)input, (double*)output,
                        info->inside, info->axis, info->outside,
                        info->insideStride, info->axisStride);
        checkKernelErrors;
        return;
    }
    switch (bytes) {
        case 4:
            PACKCOMMON<<<cores, threadNumbers>>>((const float*)input, (float*)output, 
                        info->inside, info->axis, info->outside, 
                        info->insideStride, info->axisStride);
            break;
        case 2:
            PACKCOMMON<<<cores, threadNumbers>>>((const half*)input, (half*)output, 
                        info->inside, info->axis, info->outside, 
                        info->insideStride, info->axisStride);
            break;
        case 1:
            PACKCOMMON<<<cores, threadNumbers>>>((const int8_t*)input, (int8_t*)output, 
                        info->inside, info->axis, info->outside, 
                        info->insideStride, info->axisStride);
            break;
        default:
            break;
    }
}
void UnpackBuffer(void* output, const void* input, const PackInfo* info, int bytes, CUDARuntime* runtime) {
    auto& prop = runtime->prop();
    int cores = prop.multiProcessorCount;
    int threadNumbers = prop.maxThreadsPerBlock;

    if (info->axis % 4 == 0 && info->axisStride == 1 && info->insideStride == info->axis) {
        int axis_pack = UP_DIV(info->axis, PACK_NUMBER) * PACK_NUMBER / 4;
        DivModFast is(info->inside);
        DivModFast cs(axis_pack);
        const int maxCount = info->inside * axis_pack * info->outside;
        int block_num = runtime->blocks_num(maxCount);
        int block_size = runtime->threads_num();
        int axisAlign = UP_DIV(info->axis / 4, PACK_NUMBER / 4) * PACK_NUMBER / 4;;
        if(bytes == 4) {        
            UNPACKCOMMON_4<<<block_num, block_size>>>((const int4*)input, (int4*)output, 
                            maxCount, info->inside, info->axis / 4, info->outside,
                            info->insideStride / 4, info->axisStride, axisAlign, is, cs);
            checkKernelErrors;
            return;
        }        
        if(bytes == 2) {
            UNPACKCOMMON_4<<<block_num, block_size>>>((const int2*)input, (int2*)output,
                        maxCount, info->inside, info->axis / 4, info->outside,
                        info->insideStride / 4, info->axisStride, axisAlign, is, cs);
            checkKernelErrors;
            return;
        }    
    }
    //printf("unpack size:%d %d %d, stride:%d %d, %p %p\n", info->outside, info->axis, info->inside, info->axisStride, info->insideStride, input, output); 

    if (info->axis % 16 == 0 && info->inside % 32 == 0 && info->insideStride == 1 && info->axisStride == info->inside && bytes == 2) {
        int thread=128;
        int block=info->axis/16 * info->inside/32 * info->outside;
        UNPACKCOMMON_REARRANGE_half_4<<<block, thread>>>((const double*)input, (double*)output,
                        info->inside, info->axis, info->outside,
                        info->insideStride, info->axisStride);
	checkKernelErrors;	
	return;
    }
    switch (bytes) {
        case 4:
            UNPACKCOMMON<<<cores, threadNumbers>>>((const float*)input, (float*)output, 
                        info->inside, info->axis, info->outside, 
                        info->insideStride, info->axisStride);
            break;
        case 2:
            UNPACKCOMMON<<<cores, threadNumbers>>>((const half*)input, (half*)output, 
                        info->inside, info->axis, info->outside, 
                        info->insideStride, info->axisStride);
            break;
        case 1:
            UNPACKCOMMON<<<cores, threadNumbers>>>((const int8_t*)input, (int8_t*)output, 
                        info->inside, info->axis, info->outside, 
                        info->insideStride, info->axisStride);
            break;
        default:
            break;
    }
}

void PackFP32ToFP16(void* output, const void* input, const PackInfo* info, CUDARuntime* runtime) {
    auto& prop = runtime->prop();
    int cores = prop.multiProcessorCount;
    int threadNumbers = prop.maxThreadsPerBlock;
    PACKCOMMON<<<cores, threadNumbers>>>((const float*)input, (half*)output, 
                info->inside, info->axis, info->outside, 
                info->insideStride, info->axisStride);
}
void PackFP16ToFP32(void* output, const void* input, const PackInfo* info, CUDARuntime* runtime) {
    auto& prop = runtime->prop();
    int cores = prop.multiProcessorCount;
    int threadNumbers = prop.maxThreadsPerBlock;
    PACKCOMMON<<<cores, threadNumbers>>>((const half*)input, (float*)output, 
                info->inside, info->axis, info->outside, 
                info->insideStride, info->axisStride);
}

void UnpackFP16ToFP32(void* output, const void* input, const PackInfo* info, CUDARuntime* runtime) {
    auto& prop = runtime->prop();
    int cores = prop.multiProcessorCount;
    int threadNumbers = prop.maxThreadsPerBlock;
    UNPACKCOMMON<<<cores, threadNumbers>>>((const half*)input, (float*)output, 
                    info->inside, info->axis, info->outside, 
                    info->insideStride, info->axisStride);
}
void UnpackFP32ToFP16(void* output, const void* input, const PackInfo* info, CUDARuntime* runtime) {
    auto& prop = runtime->prop();
    int cores = prop.multiProcessorCount;
    int threadNumbers = prop.maxThreadsPerBlock;
    UNPACKCOMMON<<<cores, threadNumbers>>>((const float*)input, (half*)output, 
                    info->inside, info->axis, info->outside, 
                    info->insideStride, info->axisStride);
}



template<typename T>
__global__ void TRANSPOSE(const T *input, T *output, const TransposeParam* param) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < param->total) {
        int x = i % param->dims[0];
        int tmp = i / param->dims[0];
        int y = tmp % param->dims[1];
        int z = tmp / param->dims[1];
        int srcOffset = param->srcStride * z + y + x * param->dims[2];
        int dstOffset = param->dstStride * z + x + y * param->dims[3];
        output[dstOffset] = input[srcOffset];
    }
}
#define LOCAL_DIM 8

template <typename T>
__global__ void TRANSPOSE_LOCAL(const T* input, T *output, const TransposeParam* param) {
    __shared__ T localM[LOCAL_DIM][LOCAL_DIM + 1];
    int num = blockIdx.z;
    for (int n = num; n < param->size; n += gridDim.z) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x < param->dims[0] && y < param->dims[1]) {
            int offset                      = n * param->srcStride + x * param->dims[2] + y;
            localM[threadIdx.y][threadIdx.x] = input[offset];
        }
        __syncthreads();
        x = blockIdx.y * blockDim.y + threadIdx.x;
        y = blockIdx.x * blockDim.x + threadIdx.y;
        if (x < param->dims[1] && y < param->dims[0]) {
            int offset = n * param->dstStride + x * param->dims[3] + y;
            output[offset] = localM[threadIdx.x][threadIdx.y];
        }
    }
}

void Transpose(uint8_t* output, const uint8_t* input, const TransposeParam* cpuParam, const TransposeParam* gpuRegion, int bytes, CUDARuntime* runtime) {
    int count = cpuParam->total;
    int block_num = runtime->blocks_num(count);
    int threads_num = runtime->threads_num();
    auto out = output + bytes * cpuParam->dstOffset;
    auto inp = input + bytes * cpuParam->srcOffset;
    if (runtime->prop().maxThreadsPerBlock >= LOCAL_DIM * LOCAL_DIM && (cpuParam->dims[0] >= LOCAL_DIM || cpuParam->dims[1] >= LOCAL_DIM)) {
        dim3 localSize(LOCAL_DIM, LOCAL_DIM, 1);
        //printf("%d, %d - %d, %d - %d\n", cpuParam->size, cpuParam->dims[0], cpuParam->dims[1], cpuParam->dims[2], cpuParam->dims[3]);
        int globalZ = ALIMIN(runtime->prop().multiProcessorCount, cpuParam->size);
        dim3 globalSize(UP_DIV(cpuParam->dims[0], LOCAL_DIM), UP_DIV(cpuParam->dims[1], LOCAL_DIM), globalZ);
        switch (bytes) {
            case 4:
                TRANSPOSE_LOCAL<<<globalSize, localSize>>>((const float *)inp, (float *)out, gpuRegion);
                break;
            case 2:
                TRANSPOSE_LOCAL<<<globalSize, localSize>>>((const half *)inp, (half *)out, gpuRegion);
                break;
            case 1:
                TRANSPOSE_LOCAL<<<globalSize, localSize>>>((const int8_t *)inp, (int8_t *)out, gpuRegion);
                break;
            default:
                break;
        }
        return;
    }
    switch (bytes) {
        case 4:
            TRANSPOSE<<<block_num, threads_num>>>((int*)inp, (int*)out, gpuRegion);
            break;
        case 2:
            TRANSPOSE<<<block_num, threads_num>>>((int16_t*)inp, (int16_t*)out, gpuRegion);
            break;
        case 1:
            TRANSPOSE<<<block_num, threads_num>>>((int8_t*)inp, (int8_t*)out, gpuRegion);
            break;
        default:
            break;
    }
}

// for the following transpose kernels:
// maxCount is num of threads i.e., num of elements of output format
// inChannelPack is num of channel pack of input format
// divOutChannelPack is Div for channel pack of output format

// copy kernel
template<typename T0, typename T1>
__global__ void NCHW_2_NCHW(const T0* input,
                            T1* output,
                            const int maxCount
) {
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        output[index] = (T1)input[index];
    }
}

// NHWC NCHW
template<typename T0, typename T1>
__global__ void NHWC_2_NCHW(const T0* input,
                            T1* output,
                            const int maxCount,
                            const int channel, // redundant parameter
                            const int area,
                            const int inChannelPack,
                            DivModFast divOutChannelPack,
                            DivModFast divArea
) {
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int area_idx, temp, chnl_idx, batch_idx;
        divArea.divmod(index, temp, area_idx);
        divOutChannelPack.divmod(temp, batch_idx, chnl_idx);

        int src_offset = (batch_idx * area + area_idx) * inChannelPack+ chnl_idx;
        output[index] = (T1)input[src_offset];
    }
}

// NHWC8_2_NCHW
template<typename T0, typename T1>
__global__ void NHWC8_2_NCHW(const T0* input,
                             T1* output,
                             const int maxCount,
                             const int channel, // redundant parameter
                             const int area,
                             const int inChannelPack,
                             DivModFast divOutChannelPack,
                             DivModFast divArea
) {
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int area_idx, temp, chnl_idx, batch_idx;
        divArea.divmod(index, temp, area_idx);
        divOutChannelPack.divmod(temp, batch_idx, chnl_idx);

        int src_offset = (batch_idx * area + area_idx) * inChannelPack + chnl_idx;
        output[index] = (T1)input[src_offset];
    }
}

// C4NHW4_2_NCHW
template<typename T0, typename T1>
__global__ void C4NHW4_2_NCHW(const T0* input,
                             T1* output,
                             const int maxCount,
                             const int channel,
                             const int area,
                             const int inChannelPack, // redundant parameter
                             DivModFast divOutChannelPack,
                             DivModFast divArea
) {
    const int batch = (maxCount / channel) / area;
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int area_idx, temp, chnl_idx, batch_idx;
        divArea.divmod(index, temp, area_idx);
        divOutChannelPack.divmod(temp, batch_idx, chnl_idx);

        int c4_idx = chnl_idx >> 2;
        int cL_idx = chnl_idx & 3;
        int src_offset = ((c4_idx * batch + batch_idx) * area + area_idx) * 4 + cL_idx;
        output[index] = (T1)input[src_offset];
    }
}

// NCHW NHWC
template<typename T0, typename T1>
__global__ void NCHW_2_NHWC(const T0* input,
                            T1* output,
                            const int maxCount,
                            const int channel, // redundant parameter
                            const int area,
                            const int inChannelPack,
                            DivModFast divOutChannelPack,
                            DivModFast divArea
) {
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int area_idx, temp, chnl_idx, batch_idx;
        divOutChannelPack.divmod(index, temp, chnl_idx);
        divArea.divmod(temp, batch_idx, area_idx);

        int src_offset = (batch_idx * inChannelPack + chnl_idx) * area + area_idx;
        output[index] = (T1)input[src_offset];
    }
}

// NHWC8 NHWC
template<typename T0, typename T1>
__global__ void NHWC8_2_NHWC(const T0* input,
                             T1* output,
                             const int maxCount,
                             const int channel, // redundant parameter
                             const int area,
                             const int inChannelPack,
                             DivModFast divOutChannelPack,
                             DivModFast divArea
) {
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int area_idx, temp, chnl_idx, batch_idx;
        divOutChannelPack.divmod(index, temp, chnl_idx);
        divArea.divmod(temp, batch_idx, area_idx);

        int src_offset = (batch_idx * area + area_idx) * inChannelPack + chnl_idx;
        output[index] = (T1)input[src_offset];
    }
}

// C4NHW4 NHWC
template<typename T0, typename T1>
__global__ void C4NHW4_2_NHWC(const T0* input,
                             T1* output,
                             const int maxCount,
                             const int channel,
                             const int area,
                             const int inChannelPack, // redundant parameter
                             DivModFast divOutChannelPack,
                             DivModFast divArea
) {
    const int batch = (maxCount / channel) / area;
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int area_idx, temp, chnl_idx, batch_idx;
        divOutChannelPack.divmod(index, temp, chnl_idx);
        divArea.divmod(temp, batch_idx, area_idx);

        int c4_idx = chnl_idx >> 2;
        int cL_idx = chnl_idx & 3;
        int src_offset = ((c4_idx * batch + batch_idx) * area + area_idx) * 4 + cL_idx;
        output[index] = (T1)input[src_offset];
    }
}

// NHWC NHWC8
template<typename T0, typename T1>
__global__ void NHWC_2_NHWC8(const T0* input,
                             T1* output,
                             const int maxCount,
                             const int channel,
                             const int area,
                             const int inChannelPack,
                             DivModFast divOutChannelPack,
                             DivModFast divArea
) {
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int area_idx, temp, chnl_idx, batch_idx;
        divOutChannelPack.divmod(index, temp, chnl_idx);
        divArea.divmod(temp, batch_idx, area_idx);

        if(chnl_idx >= channel) {
            output[index] = (T1)0.0f;
            continue;
        }

        int src_offset = (batch_idx * area + area_idx) * inChannelPack + chnl_idx;
        output[index] = (T1)input[src_offset];
    }
}

// NCHW NHWC8
template<typename T0, typename T1>
__global__ void NCHW_2_NHWC8(const T0* input,
                             T1* output,
                             const int maxCount,
                             const int channel,
                             const int area,
                             const int inChannelPack,
                             DivModFast divOutChannelPack,
                             DivModFast divArea
) {
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int area_idx, temp, chnl_idx, batch_idx;
        divOutChannelPack.divmod(index, temp, chnl_idx);
        divArea.divmod(temp, batch_idx, area_idx);

        if(chnl_idx >= channel) {
            output[index] = (T1)0.0f;
            continue;
        }

        int src_offset = (batch_idx * inChannelPack + chnl_idx) * area + area_idx;
        output[index] = (T1)input[src_offset];
    }
}

// C4NHW4 NHWC8
template<typename T0, typename T1>
__global__ void C4NHW4_2_NHWC8(const T0* input,
                              T1* output,
                              const int maxCount,
                              const int channel,
                              const int area,
                             const int inChannelPack, // redundant parameter
                              DivModFast divOutChannelPack,
                              DivModFast divArea
) {
    const int batch = (maxCount / (UP_DIV(channel, 8) * 8)) / area;
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int area_idx, temp, chnl_idx, batch_idx;
        divOutChannelPack.divmod(index, temp, chnl_idx);
        divArea.divmod(temp, batch_idx, area_idx);

        if(chnl_idx >= channel) {
            output[index] = (T1)0.0f;
            continue;
        }

        int c4_idx = chnl_idx >> 2;
        int cL_idx = chnl_idx & 3;
        int src_offset = ((c4_idx * batch + batch_idx) * area + area_idx) * 4 + cL_idx;
        output[index] = (T1)input[src_offset];
    }
}

// NHWC_2_C4NHW4
template<typename T0, typename T1>
__global__ void NHWC_2_C4NHW4(const T0* input,
                               T1* output,
                               const int maxCount,
                               const int channel,
                               const int area,
                               const int inChannelPack,
                               DivModFast divOutChannelPack,
                               DivModFast divArea
) {
    const int batch = (maxCount / (UP_DIV(channel, 4) * 4)) / area;
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        // arrange threads arrodring to NHWC4 format
        int area_idx, temp, chnl_idx, batch_idx;
        divOutChannelPack.divmod(index, temp, chnl_idx);
        divArea.divmod(temp, batch_idx, area_idx);

        int c4_idx = chnl_idx >> 2; // chnl_idx / 4
        int cL_idx = chnl_idx & 3; // chnl_idx % 4
        int dst_offset = ((c4_idx * batch + batch_idx) * area + area_idx) * 4 + cL_idx;
        int src_offset = (batch_idx * area + area_idx) * inChannelPack + chnl_idx;

        if (chnl_idx >= channel) {
            output[dst_offset] = (T1)0.0f;;
            continue;
        }

        output[dst_offset] = (T1)input[src_offset];
    }
}

// NCHW C4NHW4
template<typename T0, typename T1>
__global__ void NCHW_2_C4NHW4(const T0* input,
                               T1* output,
                               const int maxCount,
                               const int channel,
                               const int area,
                               const int inChannelPack,
                               DivModFast divOutChannelPack,
                               DivModFast divArea
) {
    const int batch = (maxCount / (UP_DIV(channel, 4) * 4)) / area;
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        // arrange threads arrodring to NHWC4 format
        int area_idx, temp, chnl_idx, batch_idx;
        divOutChannelPack.divmod(index, temp, chnl_idx);
        divArea.divmod(temp, batch_idx, area_idx);

        int c4_idx = chnl_idx >> 2; // chnl_idx / 4
        int cL_idx = chnl_idx & 3; // chnl_idx % 4
        int dst_offset = ((c4_idx * batch + batch_idx) * area + area_idx) * 4 + cL_idx;
        int src_offset = (batch_idx * inChannelPack + chnl_idx) * area + area_idx;

        if (chnl_idx >= channel) {
            output[dst_offset] = (T1)0.0f;;
            continue;
        }

        output[dst_offset] = (T1)input[src_offset];
    }
}

// NHWC8 C4NHW4
template<typename T0, typename T1>
__global__ void NHWC8_2_C4NHW4(const T0* input,
                               T1* output,
                               const int maxCount,
                               const int channel,
                               const int area,
                               const int inChannelPack,
                               DivModFast divOutChannelPack,
                               DivModFast divArea
) {
    const int batch = (maxCount / (UP_DIV(channel, 4) * 4)) / area;
    for(size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        // arrange threads arrodring to NHWC4 format
        int area_idx, temp, chnl_idx, batch_idx;
        divOutChannelPack.divmod(index, temp, chnl_idx);
        divArea.divmod(temp, batch_idx, area_idx);

        int c4_idx = chnl_idx >> 2; // chnl_idx / 4
        int cL_idx = chnl_idx & 3; // chnl_idx % 4
        int dst_offset = ((c4_idx * batch + batch_idx) * area + area_idx) * 4 + cL_idx;
        int src_offset = (batch_idx * area + area_idx) * inChannelPack + chnl_idx;;

        output[dst_offset] = (T1)input[src_offset];
    }
}

template<class T0, class T1>
static void insideFormatConvert(T0* input, T1* output, MNN_DATA_FORMAT srcDataFormat, MNN_DATA_FORMAT dstDataFormat, CUDARuntime* runtime, \
    const int area, const int batch, const int channel, const bool srcDevice, const bool dstDevice) {
    DivModFast d_oc(channel);
    DivModFast d_oc4(UP_DIV(channel, 4) * 4);
    DivModFast d_oc8(UP_DIV(channel, 8) * 8);
    DivModFast d_area(area);

    // NCHW NCHW
    // NHWC NHWC
    if ((srcDataFormat == MNN_DATA_FORMAT_NCHW && dstDataFormat == MNN_DATA_FORMAT_NCHW) || \
        (srcDataFormat == MNN_DATA_FORMAT_NHWC && dstDataFormat == MNN_DATA_FORMAT_NHWC)) {
        const int maxCount = batch * area * channel;
        const int block_num = runtime->blocks_num(maxCount);
        const int block_size = runtime->threads_num();
        NCHW_2_NCHW<T0, T1><<<block_num, block_size>>>(input, output, maxCount);
        checkKernelErrors;
        return;
    }

    // NC4HW4 NC4HW4
    if (srcDataFormat == MNN_DATA_FORMAT_NC4HW4 && dstDataFormat == MNN_DATA_FORMAT_NC4HW4) {
        if(!srcDevice && dstDevice) {
            const int maxCount = batch * area * UP_DIV(channel, 8) * 8;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            C4NHW4_2_NHWC8<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, UP_DIV(channel, 4) * 4, d_oc8, d_area);
            checkKernelErrors;
        } else if (srcDevice && !dstDevice) {
            const int maxCount = batch * area * UP_DIV(channel, 4) * 4;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            NHWC8_2_C4NHW4<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, UP_DIV(channel, 8) * 8, d_oc4, d_area);
            checkKernelErrors;
        } else {
            const int maxCount = batch * area * UP_DIV(channel, 8) * 8;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            NCHW_2_NCHW<T0, T1><<<block_num, block_size>>>(input, output, maxCount);
            checkKernelErrors;
        }
        return;
    }

    // NHWC NCHW
    if (srcDataFormat == MNN_DATA_FORMAT_NHWC && dstDataFormat == MNN_DATA_FORMAT_NCHW) {
        const int maxCount = batch * area * channel;
        const int block_num = runtime->blocks_num(maxCount);
        const int block_size = runtime->threads_num();
        NHWC_2_NCHW<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, channel, d_oc, d_area);
        checkKernelErrors;
        return;
    }

    // NC4HW4 NCHW
    if (srcDataFormat == MNN_DATA_FORMAT_NC4HW4 && dstDataFormat == MNN_DATA_FORMAT_NCHW) {
        if (!srcDevice) {
            const int maxCount = batch * area * channel;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            C4NHW4_2_NCHW<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, UP_DIV(channel, 4) * 4, d_oc, d_area);
            checkKernelErrors;
        } else {
            const int maxCount = batch * area * channel;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            NHWC8_2_NCHW<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, UP_DIV(channel, 8) * 8, d_oc, d_area);
            checkKernelErrors;
        }
        return;
    }

    // NCHW NHWC
    if (srcDataFormat == MNN_DATA_FORMAT_NCHW && dstDataFormat == MNN_DATA_FORMAT_NHWC) {
        const int maxCount = batch * area * channel;
        const int block_num = runtime->blocks_num(maxCount);
        const int block_size = runtime->threads_num();
        NCHW_2_NHWC<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, channel, d_oc, d_area);
        checkKernelErrors;
        return;
    }

    // NC4HWC4 NHWC
    if (srcDataFormat == MNN_DATA_FORMAT_NC4HW4 && dstDataFormat == MNN_DATA_FORMAT_NHWC) {
        if (!srcDevice) {
            const int maxCount = batch * area * channel;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            C4NHW4_2_NHWC<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, UP_DIV(channel, 4) * 4, d_oc, d_area);
            checkKernelErrors;
        } else {
            const int maxCount = batch * area * channel;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            NHWC8_2_NHWC<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, UP_DIV(channel, 8) * 8, d_oc, d_area);
            checkKernelErrors;
        }
        return;
    }

    // NCHW NC4HW4
    if(srcDataFormat == MNN_DATA_FORMAT_NCHW && dstDataFormat == MNN_DATA_FORMAT_NC4HW4) {
        if (!dstDevice) {
            const int maxCount = batch * area * UP_DIV(channel, 4) * 4;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            NCHW_2_C4NHW4<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, channel, d_oc4, d_area);
            checkKernelErrors;
        } else {
            const int maxCount = batch * area * UP_DIV(channel, 8) * 8;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            NCHW_2_NHWC8<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, channel, d_oc8, d_area);
            checkKernelErrors;
        }
        return;
    }

    // NHWC NC4HW4
    if(srcDataFormat == MNN_DATA_FORMAT_NHWC && dstDataFormat == MNN_DATA_FORMAT_NC4HW4) {
        if (!dstDevice) {
            const int maxCount = batch * area * UP_DIV(channel, 4) * 4;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            NHWC_2_C4NHW4<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, channel, d_oc4, d_area);
            checkKernelErrors;
        } else {
            const int maxCount = batch * area * UP_DIV(channel, 8) * 8;
            const int block_num = runtime->blocks_num(maxCount);
            const int block_size = runtime->threads_num();
            NHWC_2_NHWC8<T0, T1><<<block_num, block_size>>>(input, output, maxCount, channel, area, channel, d_oc8, d_area);
            checkKernelErrors;
        }
        return;
    }

    MNN_ERROR("CUDA backend doesn't support the format conversion.\n");
    MNN_ASSERT(false);
    return;
}

void FormatConvert(void* output, void* input, MNN_DATA_FORMAT srcDataFormat, MNN_DATA_FORMAT dstDataFormat, CUDARuntime* runtime, \
    const int area, const int batch, const int channel, const Tensor* srcTensor, int precision, bool srcDevice, bool dstDevice) {
    if(batch == 0 || area == 0 || channel == 0) {
        MNN_PRINT("Error: formatConvert size batch:%d - plane:%d - channel:%d, format:%d->%d, device:%d->%d\n", batch, area, channel, srcDataFormat, dstDataFormat, srcDevice, dstDevice);
        return;
    }

    bool isFp16 = (precision == 2) && (halide_type_float == srcTensor->getType().code);
    bool isBf16 = (precision == 3) && (halide_type_float == srcTensor->getType().code);

    // int8 case
    auto des = TensorUtils::getDescribe(srcTensor);
    if ((des->quantAttr.get() != nullptr && des->type == DataType_DT_INT8) || srcTensor->getType().bits == 8) {
        insideFormatConvert<int8_t, int8_t>((int8_t *)input, (int8_t *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
        return;
    }

    // FP case
    if(!srcDevice) {
        if(isFp16) {
            insideFormatConvert<float, half>((float *)input, (half *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
        } else if(isBf16) {
            #ifdef ENABLE_CUDA_BF16
            insideFormatConvert<float, __hip_bfloat16>((float *)input, (__hip_bfloat16 *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
            #endif
        } else {
            insideFormatConvert<float, float>((float *)input, (float *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
        }
    } else if(!dstDevice) {
        if(isFp16) {
            insideFormatConvert<half, float>((half *)input, (float *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
        } else if(isBf16) {
            #ifdef ENABLE_CUDA_BF16
            insideFormatConvert<__hip_bfloat16, float>((__hip_bfloat16 *)input, (float *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
            #endif
        } else {
            insideFormatConvert<float, float>((float *)input, (float *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
        }
    } else {
        if(isFp16) {
            insideFormatConvert<half, half>((half *)input, (half *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
        } else if(isBf16) {
            #ifdef ENABLE_CUDA_BF16
            insideFormatConvert<__hip_bfloat16, __hip_bfloat16>((__hip_bfloat16 *)input, (__hip_bfloat16 *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
            #endif
        } else {
            insideFormatConvert<float, float>((float *)input, (float *)output, srcDataFormat, dstDataFormat, runtime, area, batch, channel, srcDevice, dstDevice);
        }
    }
    return;
}


};
};
