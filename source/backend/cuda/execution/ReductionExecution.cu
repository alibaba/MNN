#include "ReductionExecution.hpp"
namespace MNN {
namespace CUDA {

ReductionExecution::ReductionExecution(ReductionType opType, int axis, Backend *backend) : Execution(backend) {
    mType = opType;
    mAxis = axis;
    auto staticPool = static_cast<CUDABackend*>(backend)->getStaticBufferPool();
    mParam = staticPool->alloc(sizeof(ReduceParam));
}
ReductionExecution::~ ReductionExecution() {
    auto staticPool = static_cast<CUDABackend*>(backend())->getStaticBufferPool();
    staticPool->free(mParam);
}

ErrorCode ReductionExecution::onResize(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    int inside = 1;
    int outside = 1;
    int axis = inputs[0]->length(mAxis);
    for (int i=0; i<mAxis; ++i) {
        outside *= inputs[0]->length(i);
    }
    for (int i=mAxis+1; i<inputs[0]->dimensions(); ++i) {
        inside *= inputs[0]->length(i);
    }
    mCpuParam.inside = inside;
    mCpuParam.outside = outside;
    mCpuParam.axis = axis;
    cuda_check(hipMemcpy((uint8_t*)mParam.first + mParam.second, &mCpuParam, sizeof(ReduceParam), hipMemcpyHostToDevice));
    
    //MNN_PRINT("Reduction axis_idx:%d, outside:%d, axis:%d, inside:%d\n", mAxis, outside, axis, inside);
    return NO_ERROR;
}

ErrorCode ReductionExecution::onExecute(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto input = (void*)inputs[0]->deviceId();
    auto output = (void*)outputs[0]->deviceId();
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    int inside = mCpuParam.inside;;
    int outside = mCpuParam.outside;
    int count = inside * outside;
    int block_num = runtime->blocks_num(count);
    int threads_num = runtime->threads_num();
    auto param = (ReduceParam*)((uint8_t*)mParam.first + mParam.second);
    if (inputs[0]->getType() == halide_type_of<float>()) {
        if (static_cast<CUDABackend*>(backend())->useFp16()) {
            switch (mType) {
                case ReductionType_MEAN:
                    MEAN<<<block_num, threads_num>>>((const half*)input, (half*)output, param);
                    return NO_ERROR;
                case ReductionType_SUM:
                    SUM<<<block_num, threads_num>>>((const half*)input, (half*)output, param);
                    return NO_ERROR;
                case ReductionType_MINIMUM:
                    MINIMUM<<<block_num, threads_num>>>((const half*)input, (half*)output, param);
                    return NO_ERROR;
                case ReductionType_MAXIMUM:
                    MAXIMUM<<<block_num, threads_num>>>((const half*)input, (half*)output, param);
                    return NO_ERROR;
                case ReductionType_PROD:
                    PROD<<<block_num, threads_num>>>((const half*)input, (half*)output, param);
                    return NO_ERROR;
            }
        } else {
            switch (mType) {
                case ReductionType_MEAN:
                    MEAN<<<block_num, threads_num>>>((const float*)input, (float*)output, param);
                    return NO_ERROR;
                case ReductionType_SUM:
                    SUM<<<block_num, threads_num>>>((const float*)input, (float*)output, param);
                    return NO_ERROR;
                case ReductionType_MINIMUM:
                    MINIMUM<<<block_num, threads_num>>>((const float*)input, (float*)output, param);
                    return NO_ERROR;
                case ReductionType_MAXIMUM:
                    MAXIMUM<<<block_num, threads_num>>>((const float*)input, (float*)output, param);
                    return NO_ERROR;
                case ReductionType_PROD:
                    PROD<<<block_num, threads_num>>>((const float*)input, (float*)output, param);
                    return NO_ERROR;
            }
        }
        MNN_ASSERT(false);
        return NOT_SUPPORT;
    }
    
    MNN_ASSERT(inputs[0]->getType() == halide_type_of<int32_t>());
    switch (mType) {
        case ReductionType_MEAN:
            MEAN<<<block_num, threads_num>>>((const int32_t*)input, (int32_t*)output, param);
            return NO_ERROR;
        case ReductionType_SUM:
            SUM<<<block_num, threads_num>>>((const int32_t*)input, (int32_t*)output, param);
            return NO_ERROR;
        case ReductionType_MINIMUM:
            MINIMUM<<<block_num, threads_num>>>((const int32_t*)input, (int32_t*)output, param);
            return NO_ERROR;
        case ReductionType_MAXIMUM:
            MAXIMUM<<<block_num, threads_num>>>((const int32_t*)input, (int32_t*)output, param);
            return NO_ERROR;
        case ReductionType_PROD:
            PROD<<<block_num, threads_num>>>((const int32_t*)input, (int32_t*)output, param);
            return NO_ERROR;
        case ReductionType_ANY:
            MAXIMUM<<<block_num, threads_num>>>((const int32_t*)input, (int32_t*)output, param);
            return NO_ERROR;
        case ReductionType_ALL:
            MINIMUM<<<block_num, threads_num>>>((const int32_t*)input, (int32_t*)output, param);
            return NO_ERROR;
    }
    MNN_ASSERT(false);
    return NOT_SUPPORT;
}

class ReductionCreator : public CUDABackend::Creator {
public:
    virtual Execution* onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs,
                                const MNN::Op* op, Backend* backend) const override {
        auto type = inputs[0]->getType();
        if (type.bits != 32) {
            return nullptr;
        }
        if (type.code != halide_type_float && type.code != halide_type_int) {
            return nullptr;
        }
        auto axis = op->main_as_ReductionParam()->dim()->data()[0];
        auto opType = op->main_as_ReductionParam()->operation();
        return new ReductionExecution(opType, axis, backend);
    }
};

static CUDACreatorRegister<ReductionCreator> __init(OpType_Reduction);


}
}
