#include "hip/hip_runtime.h"
//
//  CastExecution.cpp
//  MNN
//
//  Created by MNN on 2023/05/11.
//  Copyright © 2018, Alibaba Group Holding Limited
//

#include "CastExecution.hpp"
#include "core/Macro.h"
#include "core/TensorUtils.hpp"
#include "Raster.cuh"
#include "backend/cuda/core/CUDABackend.hpp"
#include "MNNCUDAFunction.cuh"
#include "MNNCUDADefine.hpp"

namespace MNN {
namespace CUDA {

template <typename T1, typename T2>
__global__ void CAST(T1 *input, T2 *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = (T2)(input[i]);
  }
  return;
}

template <typename T1, typename T2>
__global__ void CASTMIDFLOAT(T1 *input, T2 *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = (T2)((float)input[i]);
  }
  return;
}

template <typename T>
__global__ void BF162FLOAT(int16_t *input, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    float tmp;
    ((int16_t *)&tmp)[0] = 0;
    ((int16_t *)&tmp)[1] = input[i];
    output[i] = (T)tmp;
  }
}

__global__ void CASTBOOL(int32_t *input, int32_t *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    output[i] = input[i] > 0 ? 1 : 0;
  }
  return;
}

template<typename T>
__global__ void FLOAT_2_INT8_CAST(const int count,
    const T* in, 
    int8_t* out,
    const float scaleData, 
    const int8_t zeroPoint, 
    const int8_t clampMax, 
    const int8_t clampMin
) {
    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < (count); index += blockDim.x * gridDim.x) {
        float inp_0 = in[index];
        int res = __float2int_rn(inp_0 * scaleData) + zeroPoint;
        res = min(res, clampMax);
        res = max(res, clampMin);

        out[index] = res;
    }
}

template<typename T>
__global__ void INT8_2_FLOAT_CAST(const int count,
    const int8_t* in, 
    T* out,
    const float scaleData, 
    const int8_t zeroPoint
) {
    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < (count); index += blockDim.x * gridDim.x) {
        char inp_0 = in[index];
        out[index] = (T)((inp_0 - zeroPoint) * scaleData);
    }
}

template<typename T>
__global__ void FLOAT_2_INT8_CAST_PACK(const int count,
    const T* in, 
    int8_t* out,
    const float scaleData, 
    const int8_t zeroPoint, 
    const int8_t clampMax, 
    const int8_t clampMin,
    const int channelPackFloat,
    const int channels,
    DivModFast d_cp
) {
    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < (count); index += blockDim.x * gridDim.x) {
        int nhw_idx, c_idx;
        d_cp.divmod(index, nhw_idx, c_idx);
        if(c_idx >= channels) {
            out[index] = 0;
            return;
        }
        float inp_0 = in[nhw_idx * channelPackFloat + c_idx];
        int res = __float2int_rn(inp_0 * scaleData) + zeroPoint;
        res = min(res, clampMax);
        res = max(res, clampMin);

        out[index] = res;
    }
}

template<typename T>
__global__ void INT8_2_FLOAT_CAST_PACK(const int count,
    const int8_t* in, 
    T* out,
    const float scaleData, 
    const int8_t zeroPoint,
    const int channelPackInt8,
    const int channels,
    DivModFast d_cp
) {
    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < (count); index += blockDim.x * gridDim.x) {
        int nhw_idx, c_idx;
        d_cp.divmod(index, nhw_idx, c_idx);

        char inp_0 = in[nhw_idx * channelPackInt8 + c_idx];
        out[index] = (T)((inp_0 - zeroPoint) * scaleData);
    }
}

static DataType _mapDataType(DataType src) {
    if (DataType_DT_BOOL == src) {
        return DataType_DT_INT32;
    }
    if (DataType_DT_INT64 == src) {
        return DataType_DT_INT32;
    }
    if (DataType_DT_DOUBLE == src) {
        return DataType_DT_FLOAT;
    }
    return src;
}

ErrorCode CastExecution::onExecute(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    auto count = CUDABackend::realSize(inputs[0]);
    int block_num = runtime->blocks_num(count);
    int threads_num = runtime->threads_num();
    auto input = inputs[0]->deviceId();
    auto output = outputs[0]->deviceId();
    auto dstT = _mapDataType(mDst);

    const auto &inputDataType = inputs[0]->getType();
    if (inputDataType.bytes() == 4 && mDst == MNN::DataType_DT_BOOL) {
        CASTBOOL<<<block_num, threads_num>>>((int32_t*)input, (int32_t*)output, count);
        checkKernelErrors;
        return NO_ERROR;
    }
    if (inputs[0]->buffer().type == outputs[0]->buffer().type) {
        runtime->memcpy((void*)output, (void*)input, count * static_cast<CUDABackend*>(backend())->getBytes(inputs[0]), MNNMemcpyDeviceToDevice, true);
        checkKernelErrors;
        return NO_ERROR;
    }
    if (dstT == MNN::DataType_DT_INT32 && halide_type_of<int8_t>() == inputDataType) {
        CAST<<<block_num, threads_num>>>((int8_t*)input, (int32_t*)output, count);
        checkKernelErrors;
        return NO_ERROR;
    } else if (dstT == MNN::DataType_DT_UINT8 && halide_type_of<int32_t>() == inputDataType) {
        CAST<<<block_num, threads_num>>>((int32_t*)input, (uint8_t*)output, count);
        checkKernelErrors;
        return NO_ERROR;
    } else if (dstT == MNN::DataType_DT_INT32 && halide_type_of<uint8_t>() == inputDataType) {
        CAST<<<block_num, threads_num>>>((uint8_t*)input, (int32_t*)output, count);
        checkKernelErrors;
        return NO_ERROR;
    }
    if (static_cast<CUDABackend*>(backend())->useFp16()) {
        if (dstT == MNN::DataType_DT_INT32 && halide_type_of<float>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((half*)input, (int*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_FLOAT && halide_type_of<int32_t>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((int*)input, (half*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_FLOAT && halide_type_of<uint8_t>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((uint8_t*)input, (half*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_FLOAT && halide_type_of<int8_t>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((int8_t*)input, (half*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_INT8 && halide_type_of<float>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((half*)input, (int8_t*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_UINT8 && halide_type_of<float>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((half*)input, (uint8_t*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_FLOAT && halide_type_t(halide_type_float, 16) == inputDataType) {
            BF162FLOAT<<<block_num, threads_num>>>((int16_t*)input, (half*)output, count);
            checkKernelErrors;
        } else {
            MNN_PRINT("Error: CUDABackend don't support cast form %d, %d to %d\n", inputDataType.code, inputDataType.bits, dstT);
        }
    } else {
        if (dstT == MNN::DataType_DT_INT32 && halide_type_of<float>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((float*)input, (int*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_FLOAT && halide_type_of<int32_t>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((int*)input, (float*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_FLOAT && halide_type_of<uint8_t>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((uint8_t*)input, (float*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_FLOAT && halide_type_of<int8_t>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((int8_t*)input, (float*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_INT8 && halide_type_of<float>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((float*)input, (int8_t*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_UINT8 && halide_type_of<float>() == inputDataType) {
            CASTMIDFLOAT<<<block_num, threads_num>>>((float*)input, (uint8_t*)output, count);
            checkKernelErrors;
        } else if (dstT == MNN::DataType_DT_FLOAT && halide_type_t(halide_type_float, 16) == inputDataType) {
            BF162FLOAT<<<block_num, threads_num>>>((int16_t*)input, (float*)output, count);
            checkKernelErrors;
        } else {
            MNN_PRINT("Error: CUDABackend don't support cast form %d, %d to %d\n", inputDataType.code, inputDataType.bits, dstT);
        }
    }
    checkKernelErrors;
    return NO_ERROR;
}

ErrorCode CastCreator::cast(const Tensor* input, const Tensor* output, ConvertType type,
    float scale, float zero, float min, float max, Backend* bn) {
    auto runtime = static_cast<CUDABackend*>(bn)->getCUDARuntime();
    auto input_addr = (void*)input->deviceId();
    auto output_addr = (void*)output->deviceId();

    auto count = CUDABackend::realSize(input);
    // MNN_PRINT("float2int8 size:%d scale:%f\n", count, scale);
    int block_num = runtime->blocks_num(count);
    int threads_num = runtime->threads_num();
    auto sfmt    = TensorUtils::getDescribe(input)->dimensionFormat;
    auto dfmt    = TensorUtils::getDescribe(output)->dimensionFormat;
    MNN_ASSERT(sfmt == dfmt);
    if(sfmt == MNN_DATA_FORMAT_NC4HW4) {
        auto area = input->batch() * input->height() * input->width();
        auto channel = input->channel();
        auto channelPackInt8 = UP_DIV(channel, INT8_PACK_NUMBER) * INT8_PACK_NUMBER;
        auto channelPackFloat = UP_DIV(channel, PACK_NUMBER) * PACK_NUMBER;

        if (type == FlOAT_TO_INT8) {
            DivModFast cpD(channelPackInt8);
            count = area * channelPackInt8;

            scale = (scale == 0.f ? 0.f : 1.f / scale);
            if (static_cast<CUDABackend*>(bn)->useFp16()) {
                FLOAT_2_INT8_CAST_PACK<<<block_num, threads_num>>>(count, (const half *)input_addr, (int8_t *)output_addr,\
                    scale, zero, max, min, channelPackFloat, channel, cpD);
                checkKernelErrors;
            } else {
                FLOAT_2_INT8_CAST_PACK<<<block_num, threads_num>>>(count, (const float *)input_addr, (int8_t *)output_addr,\
                    scale, zero, max, min, channelPackFloat, channel, cpD);
                checkKernelErrors;
            }
            return NO_ERROR;
        }
        if (type == INT8_TO_FlOAT) {
            DivModFast cpD(channelPackFloat);
            count = area * channelPackFloat;

            if (static_cast<CUDABackend*>(bn)->useFp16()) {
                INT8_2_FLOAT_CAST_PACK<<<block_num, threads_num>>>(count, (const int8_t *)input_addr, (half *)output_addr,\
                    scale, zero, channelPackInt8, channel, cpD);
                checkKernelErrors;
            } else {
                INT8_2_FLOAT_CAST_PACK<<<block_num, threads_num>>>(count, (const int8_t *)input_addr, (float *)output_addr,\
                    scale, zero, channelPackInt8, channel, cpD);
                checkKernelErrors;
            }
            return NO_ERROR;
        }
        MNN_ERROR("CUDA Don't support NC4HW4 cast type \n");

        return NO_ERROR;
    }

    if (type == FlOAT_TO_INT8) {
        scale = (scale == 0.f ? 0.f : 1.f / scale);
        if (static_cast<CUDABackend*>(bn)->useFp16()) {
            FLOAT_2_INT8_CAST<<<block_num, threads_num>>>(count, (const half *)input_addr, (int8_t *)output_addr,\
                scale, zero, max, min);
            checkKernelErrors;
        } else {
            FLOAT_2_INT8_CAST<<<block_num, threads_num>>>(count, (const float *)input_addr, (int8_t *)output_addr,\
                scale, zero, max, min);
            checkKernelErrors;
        }
        return NO_ERROR;
    }
    if (type == INT8_TO_FlOAT) {
        if (static_cast<CUDABackend*>(bn)->useFp16()) {
            INT8_2_FLOAT_CAST<<<block_num, threads_num>>>(count, (const int8_t *)input_addr, (half *)output_addr,\
                scale, zero);
            checkKernelErrors;
        } else {
            INT8_2_FLOAT_CAST<<<block_num, threads_num>>>(count, (const int8_t *)input_addr, (float *)output_addr,\
                scale, zero);
            checkKernelErrors;
        }
        return NO_ERROR;
    }
    MNN_ERROR("CUDA Don't support cast type \n");
    return NOT_SUPPORT;
}

ErrorCode CastCreator::cast(const Tensor* input, const Tensor* output, Backend* bn, ConvertType type) {
    auto quantAttr = TensorUtils::getDescribe(input)->quantAttr;
    if (quantAttr == nullptr) {
        MNN_ERROR("No quant info for CUDA Cast srcDataType:%d\n", static_cast<CUDABackend *>(bn)->getDataType(input));
        return INVALID_VALUE;
    }
    // MNN_PRINT("quant info for Cast %d\n", static_cast<const CUDABackend*>(bn)->getDataType(input));
    auto code = cast(input, output, type, quantAttr->scale, quantAttr->zero, quantAttr->min, quantAttr->max, bn);
    if (NO_ERROR != code) {
        MNN_ERROR("Error in CUDACast\n");
        return code;
    }
    return NO_ERROR;
}


Execution* CastCreator::onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs,
                            const MNN::Op* op, Backend* backend) const{
    return new CastExecution(backend, op->main_as_CastParam()->dstT());
}

CUDACreatorRegister<CastCreator> __CastExecution(OpType_Cast);
} // namespace CUDA
} // namespace MNN
