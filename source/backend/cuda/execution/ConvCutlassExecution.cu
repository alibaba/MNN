#include "hip/hip_runtime.h"
//
//  ConvCutlassExecution.cpp
//  MNN
//
//  Created by MNN on 2020/08/22.
//  Copyright © 2018, Alibaba Group Holding Limited
//

#include "ConvCutlassExecution.hpp"
#include "Raster.cuh"
#include "ConvBaseKernel.cuh"

//#define DEBUG

namespace MNN {
namespace CUDA {

ConvCutlassExecution::Resource::Resource(Backend* bn, const MNN::Op* op) {
    mBackend = bn;
    auto runtime = static_cast<CUDABackend*>(bn)->getCUDARuntime();

    auto conv       = op->main_as_Convolution2D();
    auto common     = conv->common();

    //weight host->device
    const float* filterDataPtr = nullptr;
    int weightSize = 0;
    std::shared_ptr<ConvolutionCommon::Int8Common> quanCommon;
    ConvolutionCommon::getConvParameters(&quanCommon, conv, &filterDataPtr, &weightSize);
    auto oc = common->outputCount();

    int l = weightSize / oc;
    int h = oc;
    int lp = UP_DIV(l, 8) * 8;
    int hp = UP_DIV(h, 8) * 8;

    // Reorder weight
    {
        auto tempCacheBuffer = static_cast<CUDABackend*>(bn)->getStaticBufferPool()->alloc(weightSize * sizeof(float));
        float* cacheWeight = (float*)((uint8_t*)tempCacheBuffer.first + tempCacheBuffer.second);
        runtime->memcpy(cacheWeight, filterDataPtr, weightSize * sizeof(float), MNNMemcpyHostToDevice);
        if(static_cast<CUDABackend*>(bn)->getPrecision() == 1) {
            weightTensor.reset(Tensor::createDevice<int32_t>({lp * hp}));
        } else {
            weightTensor.reset(Tensor::createDevice<int16_t>({lp * hp}));
        }
        bn->onAcquireBuffer(weightTensor.get(), Backend::STATIC);
        mFilter = (void *)weightTensor.get()->buffer().device;

        int precision = static_cast<CUDABackend*>(bn)->getPrecision();
        if(precision == 2) {
            precision == 0;
        }
        callWeightFill((const void *)cacheWeight, (void *)mFilter, l, h, lp, hp, static_cast<CUDABackend*>(bn)->getPrecision() == 1, runtime);

        static_cast<CUDABackend*>(bn)->getStaticBufferPool()->free(tempCacheBuffer);
    }

    // Copy Bias
    {
        if(static_cast<CUDABackend*>(bn)->useFp16()) {
            auto tempBiasStorage = static_cast<CUDABackend*>(bn)->getStaticBufferPool()->alloc(conv->bias()->size()*sizeof(float));
            auto biasTemp = (float*)((uint8_t*)tempBiasStorage.first + tempBiasStorage.second);
            cuda_check(hipMemcpy(biasTemp, conv->bias()->data(), conv->bias()->size()*sizeof(float), hipMemcpyHostToDevice));

            int biasSize = conv->bias()->size();
            int hp = UP_DIV(biasSize, 8) * 8;
            biasTensor.reset(Tensor::createDevice<int16_t>({hp}));
            bn->onAcquireBuffer(biasTensor.get(), Backend::STATIC);
            mBias = (void *)biasTensor.get()->buffer().device;
            runtime->memset(mBias, 0, hp * sizeof(int16_t));

            callFloat2Half((const void*)biasTemp, (void*)mBias, hp, runtime);

            static_cast<CUDABackend*>(bn)->getStaticBufferPool()->free(tempBiasStorage);
        } else {
            int biasSize = conv->bias()->size();
            int hp = UP_DIV(biasSize, 8) * 8;
            biasTensor.reset(Tensor::createDevice<int32_t>({hp}));
            bn->onAcquireBuffer(biasTensor.get(), Backend::STATIC);
            mBias = (void *)biasTensor.get()->buffer().device;
            runtime->memset(mBias, 0, hp * sizeof(int32_t));
            cuda_check(hipMemcpy(mBias, conv->bias()->data(), conv->bias()->size()*sizeof(float), hipMemcpyHostToDevice));
        }
    }
}

ConvCutlassExecution::Resource::~Resource() {
    // Do nothing
}
ConvCutlassExecution::ConvCutlassExecution(Backend* backend, const MNN::Op* op, std::shared_ptr<Resource> res) : CutlassConvCommonExecution(backend) {
    mOp = op;
    mResource = res;
    auto runtime = static_cast<CUDABackend*>(backend)->getCUDARuntime();
    mPrecisonLevel = static_cast<CUDABackend*>(backend)->getPrecision();
    mFp16Infer = (mPrecisonLevel == 2);
    mFp32Infer = (mPrecisonLevel == 1);
    mFp16Fp32MixInfer = (mPrecisonLevel == 0);
}

ConvCutlassExecution::~ConvCutlassExecution() {

}
bool ConvCutlassExecution::onClone(Backend* bn, const Op* op, Execution** dst) {
    if (!mValid) {
        return false;
    }
    if (nullptr == dst) {
        return true;
    }
    auto dstExe = new ConvCutlassExecution(bn, op, mResource);
    *dst = dstExe;
    return true;
}


ErrorCode ConvCutlassExecution::onResize(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    auto input = inputs[0], output = outputs[0];
    const int UNIT = PACK_NUMBER;
    auto convCommon = mOp->main_as_Convolution2D()->common();
    auto pads = ConvolutionCommon::convolutionPadFull(input, output, mOp->main_as_Convolution2D()->common());
    int ic = input->channel();
    auto icDiv = UP_DIV(ic, UNIT);

    mIm2ColParamter.dilateX         = convCommon->dilateX();
    mIm2ColParamter.dilateY         = convCommon->dilateY();
    mIm2ColParamter.strideX         = convCommon->strideX();
    mIm2ColParamter.strideY         = convCommon->strideY();
    mIm2ColParamter.icDiv4          = icDiv;
    mIm2ColParamter.kernelX         = convCommon->kernelX();
    mIm2ColParamter.kernelY         = convCommon->kernelY();
    mIm2ColParamter.padX = std::get<0>(pads);
    mIm2ColParamter.padY = std::get<1>(pads);

    mIm2ColParamter.ih = input->height();
    mIm2ColParamter.iw = input->width();
    mIm2ColParamter.oh = output->height();
    mIm2ColParamter.ow = output->width();
    mIm2ColParamter.srcZStep = input->height() * input->width() * UNIT * input->batch();
    mIm2ColParamter.srcYStep = input->width() * UNIT;
    mIm2ColParamter.packCUnit = UNIT;

    mActivationType = convCommon->relu() ? 1 : convCommon->relu6() ? 2 : 0;

    //MNN_PRINT("conv size:%d-%d, %d-%d-%d, %d-%d-%d\n", mIm2ColParamter.kernelX, mIm2ColParamter.strideX, input->height(), input->width(), input->channel(), output->height(), output->width(), output->channel());
    int e = output->height() * output->width() * output->batch();
    int l = ic * mIm2ColParamter.kernelX * mIm2ColParamter.kernelY;
    int h = output->channel();
    mGemmInfo.elh[0] = e;
    mGemmInfo.elh[1] = l;
    mGemmInfo.elh[2] = h;
    mGemmInfo.elhPad[0] = UP_DIV(e, 8) * 8;
    mGemmInfo.elhPad[1] = UP_DIV(l, 8) * 8;
    mGemmInfo.elhPad[2] = UP_DIV(h, 8) * 8;

    //MNN_PRINT("Activate:%d \n", mActivationType);
    //MNN_PRINT("Im2Col：%d-%d-%d temp size:%zu!!!\n\n",output->width(), ic, mIm2ColParamter.kernelX, (size_t)sizeof(__half) * mMatMulParam.elhPack[0] * mMatMulParam.elhPack[1] * MATMULPACK * MATMULPACK);
    // When Im2Col memory size big than 2GB
    if(0){//(size_t)mGemmInfo.elh[0] * (size_t)mGemmInfo.elh[1] > 1024*1024*1024 && mIm2ColParamter.kernelX > 1 && mIm2ColParamter.kernelY > 1) {
        //printf("need im2col in block\n");
        mIsBlock = true;
        mBlockNum = 16;
        mGemmInfo.elh[0] = UP_DIV(mGemmInfo.elh[0], mBlockNum);
    }

    mIsConv1x1S1D1P0 = (mIm2ColParamter.kernelX == 1 && mIm2ColParamter.kernelY == 1 && \
                        mIm2ColParamter.strideX == 1 && mIm2ColParamter.strideY == 1 && \
                        mIm2ColParamter.dilateX == 1 && mIm2ColParamter.dilateY == 1 && \
                        mIm2ColParamter.padX == 0 && mIm2ColParamter.padY == 0);
    mNeedIm2Col = !(mIsConv1x1S1D1P0 && (mFp16Infer || mFp32Infer));

    auto pool = static_cast<CUDABackend*>(backend())->getBufferPool();
    if(mNeedIm2Col) {
        size_t im2colBytes = 2;
        // Only when fp32 Im2Col convert to fp32, Fp16Fp32Mix Im2Col convert to fp16
        if(mFp32Infer) {
            im2colBytes = 4;
        }
        auto buffer = pool->alloc(im2colBytes * (size_t)mGemmInfo.elh[0] * (size_t)mGemmInfo.elhPad[1]);
        mIm2ColBuffer = (void*)((uint8_t*)buffer.first + buffer.second);
        pool->free(buffer);
    }


    mFilterAddr = mResource->mFilter;
    mBiasAddr   = mResource->mBias;
    mBackendPtr = mResource->mBackend;

    // Call from different function
    if(mFp32Infer){
        return callCutlassGemmCudaCoreFloat32(inputs, outputs);
    } 

    mGpuComputeCap = runtime->compute_capability();
    //MNN_PRINT("Gpu smArch is sm_%d\n", mGpuComputeCap);
    if(mGpuComputeCap < 70) {
        return callCutlassGemmCudaCoreFloat16(inputs, outputs);
    } else if(mGpuComputeCap < 75) {
        return callCutlassGemmTensorCore884(inputs, outputs);
    }

    return callCutlassGemmTensorCore(inputs, outputs);
}

ErrorCode ConvCutlassExecution::onExecute(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    //MNN_PRINT("cuda convSingleInput onExecute in, inputsize:%d %d\n", (int)inputs.size(), workspace_size_);
    MNN_ASSERT(inputs.size() == 1);
    MNN_ASSERT(outputs.size() == 1);
    auto input = inputs[0];
    auto output = outputs[0];

    //printf("convcutlass:%p %p\n", input->deviceId(), output->deviceId());
    //MNN_PRINT("cutlass hw:%d-%d\n", input->height(), input->width());
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    const void *input_addr = (const void*)inputs[0]->deviceId();
    const void *filter_addr = mResource->mFilter;
    const void *bias_addr = mResource->mBias;
    auto bn = backend();
    void *output_addr = (void*)outputs[0]->deviceId();

    const int sw = mIm2ColParamter.strideX;
    const int sh = mIm2ColParamter.strideY;
    const int dw = mIm2ColParamter.dilateX;
    const int dh = mIm2ColParamter.dilateY;
    const int pw = mIm2ColParamter.padX;
    const int ph = mIm2ColParamter.padY;
    const int icDiv4 = mIm2ColParamter.icDiv4;
    const int iw = mIm2ColParamter.iw;
    const int ih = mIm2ColParamter.ih;

    //printf("%d-%d-%d-%d-%d, %d-%d\n", cpuIm2Col->icDiv4, cpuIm2Col->ih, cpuIm2Col->iw, cpuIm2Col->oh, cpuIm2Col->ow, eAlign, lAlign);
    // Im2col in Block
    for(int block_idx = 0; block_idx < mBlockNum; block_idx++) {
        if(mIsConv1x1S1D1P0 && mFp16Fp32MixInfer) {
            size_t maxCount = mGemmInfo.elh[0] * mGemmInfo.elhPad[1];
            callFloat2Half(input_addr, mIm2ColBuffer, maxCount, runtime);
        } else if (mNeedIm2Col) {

            callIm2ColPack((const void *)input_addr, (void *)mIm2ColBuffer, &mIm2ColParamter, mGemmInfo.elh[0], mGemmInfo.elh[1], \
                mGemmInfo.elhPad[0], mGemmInfo.elhPad[1], mPrecisonLevel, runtime);
        }
    }

    // Run cutlass gemm forward
    return runCutlassGemmFunc();
}


}// namespace CUDA
}// namespace MNN