#include "hip/hip_runtime.h"
//
//  SelectExecution.cpp
//  MNN
//
//  Created by MNN on 2021/12/08.
//  Copyright © 2018, Alibaba Group Holding Limited
//

#include "SelectExecution.hpp"
#include "core/Macro.h"
#include <hip/hip_runtime.h>

namespace MNN {
namespace CUDA {
#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

template<typename T>
__global__ void SELECT(const int size, const int* input0, const T* input1, const T* input2,
    int s1, int s2, T* output) {
    CUDA_KERNEL_LOOP(i, size) {
        if (input0[i] > 0) {
            output[i] = input1[i*s1];
        } else {
            output[i] = input2[i*s2];
        }
    }
}

SelectExecution::SelectExecution(Backend* backend) : Execution(backend) {
    // Do nothing
}
ErrorCode SelectExecution::onResize(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs) {
    // Do nothing
    return NO_ERROR;
}

ErrorCode SelectExecution::onExecute(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs) {
#ifdef LOG_VERBOSE
    MNN_PRINT("start SelectExecution onExecute...");
#endif
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    auto count = CUDABackend::realSize(inputs[0]);
    auto inputS1 = CUDABackend::realSize(inputs[1]);
    auto inputS2 = CUDABackend::realSize(inputs[2]);
    int s1 = inputS1 == 1 ? 0 : 1;
    int s2 = inputS2 == 1 ? 0 : 1;
    int block_num = runtime->blocks_num(count);
    int threads_num = runtime->threads_num();
    if (static_cast<CUDABackend*>(backend())->useFp16()) {
        SELECT<<<block_num, threads_num>>>(count, (const int*)(inputs[0]->deviceId()), (const half*)(inputs[1]->deviceId()), (const half*)(inputs[2]->deviceId()), \
                                            s1, s2, (half*)outputs[0]->deviceId());
        checkKernelErrors;
    } else {
        SELECT<<<block_num, threads_num>>>(count, (const int*)(inputs[0]->deviceId()), (const float*)(inputs[1]->deviceId()), (const float*)(inputs[2]->deviceId()), \
                                            s1, s2, (float*)outputs[0]->deviceId());
        checkKernelErrors;
    }
#ifdef LOG_VERBOSE
    MNN_PRINT("end SelectExecution onExecute...");
#endif
    return NO_ERROR;
}


class SelectCreator : public CUDABackend::Creator {
public:
    virtual Execution* onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs,
                                const MNN::Op* op, Backend* backend) const override {
        return new SelectExecution(backend);
    }
};

CUDACreatorRegister<SelectCreator> __SelectExecution(OpType_Select);
} // namespace CUDA
} // namespace MNN
