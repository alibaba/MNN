#include "hip/hip_runtime.h"
#include "ConvDepthWiseExecution.hpp"
#include "core/ConvolutionCommon.hpp"
#include "Raster.cuh"
#include <float.h>
#include "MNNCUDADefine.hpp"
#include "MNNCUDAFunction.cuh"

namespace MNN {
namespace CUDA {

template<typename T>
__global__ void CONV_DW(const T* input, 
    const half* kernel, 
    const half* bias, 
    T *output, 
    const constBuffer* uConstant
) {
    float maxV = uConstant->maxValue;
    float minV = uConstant->minValue;
    int iw = uConstant->inputSize[0];
    int ih = uConstant->inputSize[1];
    int c = uConstant->channel;
    int c_p = c * PACK_NUMBER;
    int ow = uConstant->outputSize[0];
    int oh = uConstant->outputSize[1];
    int kw = uConstant->kernelSize[0];
    int kh = uConstant->kernelSize[1];
    int dw = uConstant->dilate[0];
    int dh = uConstant->dilate[1];
    int sw = uConstant->stride[0];
    int sh = uConstant->stride[1];
    int pw = uConstant->pad[0];
    int ph = uConstant->pad[1];

    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < uConstant->total; index += blockDim.x * gridDim.x) {
        int i = index / c_p;
        int oz = index % c_p;
        int ob = i / (ow * oh);
        int tmp = i % (ow * oh);
        int oy = tmp / ow;
        int ox = tmp % ow;
        
        int ix = ox * sw - pw;
        int iy = oy * sh - ph;
        float color = bias[oz];
        int fxSta = max(0, (UP_DIV(-ix, dw)));
        int fySta = max(0, (UP_DIV(-iy, dh)));
        int fxEnd = min(kw, UP_DIV(iw - ix, dw));
        int fyEnd = min(kh, UP_DIV(ih - iy, dh));
        int fx, fy, fz;
        for (fy=fySta; fy<fyEnd; ++fy) {
            int sy = fy*dh + iy;
            for (fx=fxSta; fx<fxEnd; ++fx) {
                int sx = fx*dw + ix;
                float inp = input[0
                    + sx * c_p
                    + sy * iw * c_p
                    + ob * iw * ih * c_p
                    + oz
                ];
                float ker = kernel[0
                    + fx
                    + fy * kw
                    + oz * kw * kh
                ];
                color = color + inp * ker;
            }
        }
        color = max(color, minV);
        color = min(color, maxV);

        output[0
            + ox * c_p
            + oy * ow * c_p
            + ob * ow * oh * c_p
            + oz
        ] = color;
    }
}


__global__ void CONV_DW_OPT(const float* input, const half* kernel, const half* bias, float *output, const constBuffer* uConstant,
    DivModFast d_oc,
    DivModFast d_ow,
    DivModFast d_oh
    ) {
    float maxV = uConstant->maxValue;
    float minV = uConstant->minValue;
    int iw = uConstant->inputSize[0];
    int ih = uConstant->inputSize[1];
    int kw = uConstant->kernelSize[0];
    int kh = uConstant->kernelSize[1];
    int sw = uConstant->stride[0];
    int sh = uConstant->stride[1];
    int pw = uConstant->pad[0];
    int ph = uConstant->pad[1];
    int c = uConstant->channel;
    int c_p = c * PACK_NUMBER;

    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < uConstant->total; index += blockDim.x * gridDim.x) {
        int oz, tmp2, oy, ox, tmp1, ob;
        d_oc.divmod(index, tmp1, oz);
        d_ow.divmod(tmp1, tmp2, ox);
        d_oh.divmod(tmp2, ob, oy);

        int ix = ox * sw - pw;
        int iy = oy * sh - ph;
        float color = bias[oz];
        int fxSta = max(0, -ix);
        int fySta = max(0, -iy);
        int fxEnd = min(kw, iw - ix);
        int fyEnd = min(kh, ih - iy);
        int fx, fy, fz;
        for (fy=fySta; fy<fyEnd; ++fy) {
            int sy = fy + iy;
            for (fx=fxSta; fx<fxEnd; ++fx) {
                int sx = fx + ix;
                float inp = input[0
                + sx * c_p
                + sy * iw * c_p
                + ob * iw * ih * c_p
                + oz
            ];
            float ker = kernel[0
                + fx
                + fy * kw
                + oz * kw * kh
            ];
            color = color + inp * ker;
        }
    }
    color = max(color, minV);
    color = min(color, maxV);

    output[index] = color;
}
}

static std::shared_ptr<ConvDepthWiseExecution::Resource> _makeResource(const Op* op, Backend* bn) {
    std::shared_ptr<ConvDepthWiseExecution::Resource> res(new ConvDepthWiseExecution::Resource);
    auto pool = static_cast<CUDABackend*>(bn)->getStaticBufferPool();
    auto runtime = static_cast<CUDABackend*>(bn)->getCUDARuntime();
    auto conv = op->main_as_Convolution2D();
    auto convCommon = conv->common();
    int kernelX = convCommon->kernelX();
    int kernelY = convCommon->kernelY();
    int depth = convCommon->outputCount();
    int depthC = UP_DIV(depth, PACK_NUMBER);
    res->weightTensor.reset(Tensor::createDevice<float>({kernelX * kernelY * depthC * PACK_NUMBER}));
    bool success = bn->onAcquireBuffer(res->weightTensor.get(), Backend::STATIC);
    if (!success) {
        return nullptr;
    }
    res->mFilter = (void *)res->weightTensor.get()->buffer().device;
    FuseRegion reg;
    int offset[8 * PACK_NUMBER];
    auto regionStorage = static_cast<CUDABackend*>(bn)->getStaticBufferPool()->alloc(sizeof(FuseRegion));
    auto offsetGpuStorage = static_cast<CUDABackend*>(bn)->getStaticBufferPool()->alloc(sizeof(offset));
    auto offsetGpu = (uint8_t*)offsetGpuStorage.first + offsetGpuStorage.second;
    //weight host->device
    const float* filterDataPtr = nullptr;
    int weightSize = 0;
    std::shared_ptr<ConvolutionCommon::Int8Common> quanCommon;
    ConvolutionCommon::getConvParameters(&quanCommon, conv, &filterDataPtr, &weightSize);
    auto tempWeightStorage = pool->alloc(weightSize * sizeof(float));
    auto tempWeight = (uint8_t*)tempWeightStorage.first + tempWeightStorage.second;
    cuda_check(hipMemcpy(tempWeight, filterDataPtr, weightSize*sizeof(float), hipMemcpyHostToDevice));
    reg.size[0] = 1;
    reg.size[1] = depthC * PACK_NUMBER;
    reg.size[2] = kernelY * kernelX;
    reg.srcStride[0] = 0;
    reg.srcStride[1] = kernelY * kernelX;
    reg.srcStride[2] = 1;
    reg.dstStride[0] = 0;
    reg.dstStride[1] = kernelY * kernelX;
    reg.dstStride[2] = 1;
    offset[0] = 1;
    offset[1] = depth;
    offset[2] = kernelY * kernelX;
    offset[3] = 0;
    offset[4] = 1;
    offset[5] = reg.size[1];
    offset[6] = reg.size[2];
    offset[7] = 0;
    reg.fuseNumber = 1;

    runtime->memcpy((uint8_t*)regionStorage.first + regionStorage.second, &reg, sizeof(FuseRegion), MNNMemcpyHostToDevice, true);
    runtime->memcpy(offsetGpu, offset, 8 * sizeof(int), MNNMemcpyHostToDevice, true);
    FuseRasterBlitFloatToHalf((uint8_t*)res->mFilter, (uint8_t*)tempWeight, (FuseRegion*)((uint8_t*)regionStorage.first + regionStorage.second), offsetGpu, runtime);
    pool->free(tempWeightStorage);
    res->biasTensor.reset(Tensor::createDevice<float>({depthC * PACK_NUMBER}));
    success = bn->onAcquireBuffer(res->biasTensor.get(), Backend::STATIC);
    res->mBias = (void *)res->biasTensor.get()->buffer().device;
    if (!success) {
        return nullptr;
    }
    if(conv->bias() != nullptr) {
        auto tempBiasStorage = pool->alloc(depth * sizeof(float));
        auto tempBias = (uint8_t*)tempBiasStorage.first + tempBiasStorage.second;
        cuda_check(hipMemcpy(tempBias, conv->bias()->data(), conv->bias()->size()*sizeof(float), hipMemcpyHostToDevice));
        reg.size[0] = 1;
        reg.size[1] = 1;
        reg.size[2] = depthC * PACK_NUMBER;
        reg.srcStride[0] = 0;
        reg.srcStride[1] = 0;
        reg.srcStride[2] = 1;
        reg.dstStride[0] = 0;
        reg.dstStride[1] = 0;
        reg.dstStride[2] = 1;
        offset[0] = 1;
        offset[1] = 1;
        offset[2] = conv->bias()->size();
        offset[3] = 0;
        offset[4] = 1;
        offset[5] = 1;
        offset[6] = reg.size[2];
        offset[7] = 0;
        reg.fuseNumber = 1;
        runtime->memcpy((uint8_t*)regionStorage.first + regionStorage.second, &reg, sizeof(FuseRegion), MNNMemcpyHostToDevice, true);
        runtime->memcpy(offsetGpu, offset, 8 * sizeof(int), MNNMemcpyHostToDevice, true);
        FuseRasterBlitFloatToHalf((uint8_t*)res->mBias, (uint8_t*)tempBias, (FuseRegion*)((uint8_t*)regionStorage.first + regionStorage.second), offsetGpu, runtime);
        pool->free(tempBiasStorage);
    }
    static_cast<CUDABackend*>(bn)->getStaticBufferPool()->free(regionStorage);
    static_cast<CUDABackend*>(bn)->getStaticBufferPool()->free(offsetGpuStorage);
    return res;
}

ConvDepthWiseExecution::ConvDepthWiseExecution(const Op* op, Backend* bn, std::shared_ptr<Resource> resource) : Execution(bn) {
    mOp = op;
    mResource = resource;
    auto pool = static_cast<CUDABackend*>(bn)->getStaticBufferPool();
    mConstBuffer = pool->alloc(sizeof(constBuffer));
}
ConvDepthWiseExecution::~ ConvDepthWiseExecution() {
    auto pool = static_cast<CUDABackend*>(backend())->getStaticBufferPool();
    pool->free(mConstBuffer);
}

ErrorCode ConvDepthWiseExecution::onResize(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto pad = ConvolutionCommon::convolutionPad(inputs[0], outputs[0], mOp->main_as_Convolution2D()->common());
    auto conv = mOp->main_as_Convolution2D();
    auto convCommon = mOp->main_as_Convolution2D()->common();
    int channel = inputs[0]->channel();
    int channelDiv = UP_DIV(channel, PACK_NUMBER);
    parameters.pad[0] = pad.first;
    parameters.pad[1] = pad.second;
    parameters.kernelSize[0] = convCommon->kernelX();
    parameters.kernelSize[1] = convCommon->kernelY();
    parameters.stride[0] = convCommon->strideX();
    parameters.stride[1] = convCommon->strideY();
    parameters.dilate[0] = convCommon->dilateX();
    parameters.dilate[1] = convCommon->dilateY();
    parameters.inputSize[0] = inputs[0]->width();
    parameters.inputSize[1] = inputs[0]->height();
    parameters.channel = channelDiv;
    parameters.outputSize[0] = outputs[0]->width();
    parameters.outputSize[1] = outputs[0]->height();
    parameters.batch = inputs[0]->batch();

    parameters.total = parameters.batch * parameters.outputSize[1] * parameters.outputSize[0] * parameters.channel * PACK_NUMBER;
    if (static_cast<CUDABackend*>(backend())->useFp16()) {
        // Do nothing
    } else {
        parameters.minValue = -FLT_MAX;
        parameters.maxValue = FLT_MAX;
    }
    if (convCommon->relu()) {
        parameters.minValue = 0.0f;
    }
    if (convCommon->relu6()) {
        parameters.minValue = 0.0f;
        parameters.maxValue = 6.0f;
    }

    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    runtime->memcpy((uint8_t*)mConstBuffer.first + mConstBuffer.second, &parameters, sizeof(constBuffer), MNNMemcpyHostToDevice);
    mTotalCount = parameters.total;
    //printf("%d-%d-%d-%d, %d-%d-%d-%d-%d\n", parameters.kernelSize[0], parameters.kernelSize[1], parameters.stride[0], parameters.stride[1], parameters.inputSize[0], parameters.inputSize[1], channel, parameters.outputSize[0], parameters.outputSize[1]);
    return NO_ERROR;
}

ErrorCode ConvDepthWiseExecution::onExecute(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    auto& prop = runtime->prop();
    int limitThreads = UP_DIV(mTotalCount, prop.multiProcessorCount);
    int threads_num = ALIMIN(prop.maxThreadsPerBlock/2, limitThreads);
    int block_num = prop.multiProcessorCount;
    auto constPtr = (uint8_t*)mConstBuffer.first + mConstBuffer.second;
    if (static_cast<CUDABackend*>(backend())->useFp16()) {
        if (inputs.size() == 1) {
            CONV_DW<<<block_num, threads_num>>>((const half*)inputs[0]->deviceId(), (const half*)mResource->mFilter,
                (const half*)mResource->mBias, (half*)outputs[0]->deviceId(), (const constBuffer*)(constPtr));
        }
        return NO_ERROR;
    }

    if (inputs.size() == 1) {
        // block_num = runtime->blocks_num(mTotalCount);
        // threads_num = runtime->threads_num();
        if(parameters.dilate[0] == 1 && parameters.dilate[1] == 1) {
            const int area = parameters.outputSize[0] * parameters.outputSize[1];
            DivModFast d_oc(parameters.channel * PACK_NUMBER);
            DivModFast d_ow(parameters.outputSize[0]);
            DivModFast d_oh(parameters.outputSize[1]);
            
            CONV_DW_OPT<<<block_num, threads_num>>>((const float*)inputs[0]->deviceId(), (const half*)mResource->mFilter,
                (const half*)mResource->mBias, (float*)outputs[0]->deviceId(), (const constBuffer*)(constPtr),
                d_oc, d_ow, d_oh);
        } else {
            CONV_DW<<<block_num, threads_num>>>((const float*)inputs[0]->deviceId(), (const half*)mResource->mFilter,
                (const half*)mResource->mBias, (float*)outputs[0]->deviceId(), (const constBuffer*)(constPtr));
        }
    }
    return NO_ERROR;
}

class ConvDepthWiseExecutionCreator : public CUDABackend::Creator {
public:
    virtual Execution* onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs,
                                const MNN::Op* op, Backend* backend) const override {
        if (inputs.size() > 1) {
            return nullptr;
        }
        auto res = _makeResource(op, backend);
        if (nullptr == res) {
            return nullptr;
        }
        return new ConvDepthWiseExecution(op, backend, res);
    }
};

static CUDACreatorRegister<ConvDepthWiseExecutionCreator> __init(OpType_ConvolutionDepthwise);
}
}