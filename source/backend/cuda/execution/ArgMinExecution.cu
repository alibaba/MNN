#include "hip/hip_runtime.h"
//
//  ArgMinExecution.cpp
//  MNN
//
//  Created by MNN on 2022/06/29.
//  Copyright © 2018 - 2022, Alibaba Group Holding Limited
//
#include "ArgMinExecution.hpp"
#include "core/TensorUtils.hpp"
#include <MNN/AutoTime.hpp>

namespace MNN {
namespace CUDA {

template <typename T>
__global__ void ARGMIN(const int count, const int outside, const int inside, const int dim,
                         const T *input, int *output) {

    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
        const int o = i / inside;
        const int n = i % inside;

        int* outPtr = output + inside * o;
        const T* inpPtr = input + inside * dim * o;
        int index = 0;
        T minValue = inpPtr[n + 0 * inside];
        for(int j=1; j<dim; j++) {
            T value = inpPtr[n + j * inside];
            if(minValue > value) {
                index = j;
                minValue = value;
            }
        }
        outPtr[n] = index;
    }
    return;
}
ArgMinExecution::ArgMinExecution(const Op* op, Backend *backend) : Execution(backend) {
    mOp = op;
    mAxis = mOp->main_as_ArgMax()->axis();
}

ArgMinExecution::~ArgMinExecution(){
    // Do nothing
}

ErrorCode ArgMinExecution::onResize(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto input  = inputs[0];
    auto output  = outputs[0];

    if (mAxis < 0) {
        mAxis = input->dimensions() + mAxis;
    }

    mInside = 1;
    mOutside = 1;
    for (int i=0; i<mAxis; ++i) {
        mOutside *= input->length(i);
    }
    for (int i=mAxis+1; i<input->dimensions(); ++i) {
        mInside *= input->length(i);
    }
    mDim = input->length(mAxis);

    return NO_ERROR;
}

ErrorCode ArgMinExecution::onExecute(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto runtime = static_cast<CUDABackend *>(backend())->getCUDARuntime();

    auto input = (void *)inputs[0]->deviceId();
    auto output = (void *)outputs[0]->deviceId();
    int count = mOutside * mInside;
    int block_num = runtime->blocks_num(count);
    int thread_num = runtime->threads_num();

    auto bytes = static_cast<CUDABackend*>(backend())->getBytes(inputs[0]);

    if(bytes == 4) {
        ARGMIN<<<block_num, thread_num>>>(count, mOutside, mInside, mDim, (const float*)input,(int *)output);
        checkKernelErrors;
    } else {
        ARGMIN<<<block_num, thread_num>>>(count, mOutside, mInside, mDim, (const half*)input,(int *)output);
        checkKernelErrors;
    }
    return NO_ERROR;
}

}
}
