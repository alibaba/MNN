
#include <hip/hip_runtime.h>
//
//  CutlassGemmInt8TensorCore16832.cu
//  MNN
//
//  Created by MNN on 2023/01/15.
//  Copyright © 2018, Alibaba Group Holding Limited
//
#ifdef ENABLE_CUDA_QUANT

#include "ConvInt8CutlassExecution.hpp"

namespace MNN {
namespace CUDA {
ErrorCode ConvInt8CutlassExecution::callCutlassGemmInt8TensorCore16832(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    auto input = inputs[0];
    auto output = outputs[0];
    int8_t *inputA_ptr = mNeedIm2Col ? (int8_t *)mIm2ColBuffer : (int8_t *)input->deviceId();

    int8_t clamp_max = int8_t(mResource->mClampMax);
    int8_t clamp_min = int8_t(mResource->mClampMin);
    // Split K dimension into 1 partitions
    int split_k_slices = 1;
    cutlass::gemm::GemmCoord problem_size(mGemmInfo.elh[0], mGemmInfo.elhPad[2], mGemmInfo.elhPad[1]);// m n k
    if(mActivationType == 1) {
        clamp_min = int8_t(0);
    } else if(mActivationType == 2) {
        clamp_max = int8_t(6);
        clamp_min = int8_t(0);
    }

    // printf("Gemm16832 Int8 size:%d-%d-%d\n", mGemmInfo.elh[0], mGemmInfo.elhPad[2], mGemmInfo.elhPad[1]);
    // Create a tuple of gemm kernel arguments. This is later passed as arguments to launch
    // instantiated CUTLASS kernel

    typename GemmInt8Tensor_Clamp_AlignTensor_Normal_Sm80::Arguments arguments{problem_size,  // <- problem size of matrix multiplication
                                        {inputA_ptr, mGemmInfo.elhPad[1]},  // Ptr + ldm
                                        {(int8_t *)mResource->mWeightInt8Ptr, mGemmInfo.elhPad[1]},  //  Ptr + ldm
                                        {(int8_t *)output->deviceId(), mGemmInfo.elhPad[2]},  //  Ptr + ldm
                                        {(int32_t *)mResource->mBiasInt32Ptr, 0},  //  Ptr + ldm  if ldm = 0, vector, 
                                        {(float *)mResource->mScaleFloatPtr, 0},  //  Ptr + ldm  if ldm = 0, vector, 
                                        {clamp_max, clamp_min},          // <- tuple of alpha and beta
                                        split_k_slices};        // <- k-dimension split factor
    size_t workspace_size = GemmInt8Tensor_Clamp_AlignTensor_Normal_Sm80::get_workspace_size(arguments);

    if(workspace_size != 0) {
        workspaceTensor.reset(Tensor::createDevice<int8_t>({(int)workspace_size}));
        mResource->mBackend->onAcquireBuffer(workspaceTensor.get(), Backend::STATIC);
        mWorkspace = (void *)workspaceTensor.get()->buffer().device;
    }

    // Check the problem size is supported or not 
    // cutlass::Status status = mGemmInt8Clamp.can_implement(arguments);
    // cutlass_check(status);

    // Initialize CUTLASS kernel with arguments and workspace pointer
    cutlass::Status status = mGemmInt8ClampNormalSm80.initialize(arguments, (uint8_t *)mWorkspace);
    cutlass_check(status);

    return NO_ERROR;
}

}
}
#endif