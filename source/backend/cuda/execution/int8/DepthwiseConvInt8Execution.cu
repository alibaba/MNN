#include "hip/hip_runtime.h"
//
//  DepthwiseConvInt8Execution.cpp
//  MNN
//
//  Created by MNN on 2023/01/15.
//  Copyright © 2018, Alibaba Group Holding Limited
//

#include "DepthwiseConvInt8Execution.hpp"
#include "../Raster.cuh"
#include "../MNNCUDADefine.hpp"
#include "../MNNCUDAFunction.cuh"
#include <sm_61_intrinsics.h>

namespace MNN {
namespace CUDA {

__inline__ __device__
int32_t vecDot(char4 inp0, char4 inp1, int32_t val)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610))
    return __dp4a(inp0, inp1, val);
#else
    int32_t res = val;
    res += inp0.x * inp1.x;
    res += inp0.y * inp1.y;
    res += inp0.z * inp1.z;
    res += inp0.w * inp1.w;
    return res;
#endif
}

__global__ void CONV_DW_INT8_(const int8_t* input, 
    const int8_t* kernel, 
    const int32_t* bias, 
    const float*  scale,
    int8_t *output, 
    const int8_t maxV,
    const int8_t minV,
    const int iw,
    const int ih,
    const int c,
    const int c_p,
    const int ow,
    const int oh,
    const int kw,
    const int kh,
    const int k_p,
    const int dw,
    const int dh,
    const int sw,
    const int sh,
    const int pw,
    const int ph,
    const int total,
    DivModFast d_oc,
    DivModFast d_ow,
    DivModFast d_oh
) {

    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < total/4; index += blockDim.x * gridDim.x) {
        int oz_4, tmp2, oy, ox, tmp1, ob;
        d_oc.divmod(index, tmp1, oz_4);
        d_ow.divmod(tmp1, tmp2, ox);
        d_oh.divmod(tmp2, ob, oy);
        
        int oz = oz_4 << 2;
        int ix = ox * sw - pw;
        int iy = oy * sh - ph;

        int4 bias4 = ((int4 *)(bias + oz))[0];
        float color0 = (float)bias4.x;
        float color1 = (float)bias4.y;
        float color2 = (float)bias4.z;
        float color3 = (float)bias4.w;

        int fxSta = max(0, (UP_DIV(-ix, dw)));
        int fySta = max(0, (UP_DIV(-iy, dh)));
        int fxEnd = min(kw, UP_DIV(iw - ix, dw));
        int fyEnd = min(kh, UP_DIV(ih - iy, dh));
        int fx, fy, fz;
        for (fy=fySta; fy<fyEnd; ++fy) {
            int sy = fy*dh + iy;
            for (fx=fxSta; fx<fxEnd; ++fx) {
                int sx = fx*dw + ix;
                int src_offset = ((ob * ih + sy) * iw + sx) * c_p + oz;

                char4 inp4 = ((char4 *)(input + src_offset))[0];
                char4 ker4 = ((char4 *)(kernel + (fy * kw + fx) * c_p + oz))[0];;

                color0 = color0 + (float)inp4.x * (float)ker4.x;
                color1 = color1 + (float)inp4.y * (float)ker4.y;
                color2 = color2 + (float)inp4.z * (float)ker4.z;
                color3 = color3 + (float)inp4.w * (float)ker4.w;

            }
        }

        float4 scale4 = ((float4 *)(scale + oz))[0];
        color0 = color0 * scale4.x;
        color1 = color1 * scale4.y;
        color2 = color2 * scale4.z;
        color3 = color3 * scale4.w;

        color0 = max(color0, (float)minV);
        color0 = min(color0, (float)maxV);

        color1 = max(color1, (float)minV);
        color1 = min(color1, (float)maxV);

        color2 = max(color2, (float)minV);
        color2 = min(color2, (float)maxV);

        color3 = max(color3, (float)minV);
        color3 = min(color3, (float)maxV);

        int dst_offset = ((ob * oh + oy) * ow + ox) * c_p + oz;

        char4 res = make_char4(__float2int_rn(color0), __float2int_rn(color1), __float2int_rn(color2), __float2int_rn(color3));
        ((char4*)(output + dst_offset))[0] = res;
    }
}


__global__ void CONV_DW3x3S1_INT8_OPT(const int8_t* input, 
    const int8_t* kernel, 
    const int32_t* bias, 
    const float*  scale,
    int8_t *output, 
    const int8_t maxV,
    const int8_t minV,
    const int iw,
    const int ih,
    const int c,
    const int c_p,
    const int ow,
    const int oh,
    const int kw,
    const int kh,
    const int k_p,
    const int dw,
    const int dh,
    const int sw,
    const int sh,
    const int pw,
    const int ph,
    const int total,
    DivModFast d_oc,
    DivModFast d_ow,
    DivModFast d_oh
) {

    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < total/8; index += blockDim.x * gridDim.x) {
        int oz_4, tmp2, oy, ox_2, tmp1, ob;
        d_oc.divmod(index, tmp1, oz_4);
        d_ow.divmod(tmp1, tmp2, ox_2);
        d_oh.divmod(tmp2, ob, oy);
        
        int ox = ox_2 << 1;
        int oz = oz_4 << 2;
        int ix = ox - 1;
        int iy = oy - 1;

        int4 bias4 = ((int4 *)(bias + oz))[0];
        float color0_0 = (float)bias4.x;
        float color0_1 = color0_0;
        float color1_0 = (float)bias4.y;
        float color1_1 = color1_0;
        float color2_0 = (float)bias4.z;
        float color2_1 = color2_0;
        float color3_0 = (float)bias4.w;
        float color3_1 = color3_0;

        char4 zero4 = make_char4(0, 0, 0, 0);
        char4 inp4[12], ker4[3][3];
        for(int j=0; j<3; j++) {
            if(iy < 0 && j==0) {
                for(int i=0; i<4; i++) {
                    inp4[i] = zero4;
                }
                continue;
            }
            if(iy+2 > ih-1 && j==2) {
                for(int i=0; i<4; i++) {
                    inp4[8+i] = zero4;
                }
                continue;
            }

            for(int i=0; i<4; i++) {
                if(ix < 0 && i==0) {
                    for(int j=0; j<3; j++) {
                        inp4[4*j+0] = zero4;
                    }
                    continue;
                }
                if(ix+3 > iw-1 && i==3) {
                    for(int j=0; j<3; j++) {
                        inp4[4*j+3] = zero4;
                    }
                    continue;
                }
                int src_offset = ((ob * ih + iy+j) * iw + ix+i) * c_p + oz;
                inp4[4*j+i] = ((char4 *)(input + src_offset))[0];
            }
        }

        for(int j=0; j<3; j++) {
            for(int i=0; i<3; i++) {
                ker4[j][i] = ((char4 *)(kernel + (j * 3 + i) * c_p + oz))[0];// kernel[(j * 3 + i) * c_p + oz];
            }
        }

        // 1st channel
        char4 tmp0_inp4 = make_char4(inp4[0].x, inp4[1].x, inp4[2].x, inp4[4].x);
        char4 tmp1_inp4 = make_char4(inp4[1].x, inp4[2].x, inp4[3].x, inp4[5].x);
        char4 tmp_ker4 = make_char4(ker4[0][0].x, ker4[0][1].x, ker4[0][2].x, ker4[1][0].x);
        int32_t tmp0_res = vecDot(tmp0_inp4, tmp_ker4, 0);
        int32_t tmp1_res = vecDot(tmp1_inp4, tmp_ker4, 0);

        tmp0_inp4 = make_char4(inp4[5].x, inp4[6].x, inp4[8].x, inp4[9].x);
        tmp1_inp4 = make_char4(inp4[6].x, inp4[7].x, inp4[9].x, inp4[10].x);
        tmp_ker4 = make_char4(ker4[1][1].x, ker4[1][2].x, ker4[2][0].x, ker4[2][1].x);
        tmp0_res += vecDot(tmp0_inp4, tmp_ker4, 0);
        tmp1_res += vecDot(tmp1_inp4, tmp_ker4, 0);

        tmp0_res += inp4[10].x * ker4[2][2].x;
        tmp1_res += inp4[11].x * ker4[2][2].x;

        color0_0 += (float)tmp0_res;
        color0_1 += (float)tmp1_res;

        // 2nd channel
        tmp0_inp4 = make_char4(inp4[0].y, inp4[1].y, inp4[2].y, inp4[4].y);
        tmp1_inp4 = make_char4(inp4[1].y, inp4[2].y, inp4[3].y, inp4[5].y);
        tmp_ker4 = make_char4(ker4[0][0].y, ker4[0][1].y, ker4[0][2].y, ker4[1][0].y);
        tmp0_res = vecDot(tmp0_inp4, tmp_ker4, 0);
        tmp1_res = vecDot(tmp1_inp4, tmp_ker4, 0);

        tmp0_inp4 = make_char4(inp4[5].y, inp4[6].y, inp4[8].y, inp4[9].y);
        tmp1_inp4 = make_char4(inp4[6].y, inp4[7].y, inp4[9].y, inp4[10].y);
        tmp_ker4 = make_char4(ker4[1][1].y, ker4[1][2].y, ker4[2][0].y, ker4[2][1].y);
        tmp0_res += vecDot(tmp0_inp4, tmp_ker4, 0);
        tmp1_res += vecDot(tmp1_inp4, tmp_ker4, 0);

        tmp0_res += inp4[10].y * ker4[2][2].y;
        tmp1_res += inp4[11].y * ker4[2][2].y;

        color1_0 += (float)tmp0_res;
        color1_1 += (float)tmp1_res;


        // 3rd channel
        tmp0_inp4 = make_char4(inp4[0].z, inp4[1].z, inp4[2].z, inp4[4].z);
        tmp1_inp4 = make_char4(inp4[1].z, inp4[2].z, inp4[3].z, inp4[5].z);
        tmp_ker4 = make_char4(ker4[0][0].z, ker4[0][1].z, ker4[0][2].z, ker4[1][0].z);
        tmp0_res = vecDot(tmp0_inp4, tmp_ker4, 0);
        tmp1_res = vecDot(tmp1_inp4, tmp_ker4, 0);

        tmp0_inp4 = make_char4(inp4[5].z, inp4[6].z, inp4[8].z, inp4[9].z);
        tmp1_inp4 = make_char4(inp4[6].z, inp4[7].z, inp4[9].z, inp4[10].z);
        tmp_ker4 = make_char4(ker4[1][1].z, ker4[1][2].z, ker4[2][0].z, ker4[2][1].z);
        tmp0_res += vecDot(tmp0_inp4, tmp_ker4, 0);
        tmp1_res += vecDot(tmp1_inp4, tmp_ker4, 0);

        tmp0_res += inp4[10].z * ker4[2][2].z;
        tmp1_res += inp4[11].z * ker4[2][2].z;

        color2_0 += (float)tmp0_res;
        color2_1 += (float)tmp1_res;


        // 4th channel
        tmp0_inp4 = make_char4(inp4[0].w, inp4[1].w, inp4[2].w, inp4[4].w);
        tmp1_inp4 = make_char4(inp4[1].w, inp4[2].w, inp4[3].w, inp4[5].w);
        tmp_ker4 = make_char4(ker4[0][0].w, ker4[0][1].w, ker4[0][2].w, ker4[1][0].w);
        tmp0_res = vecDot(tmp0_inp4, tmp_ker4, 0);
        tmp1_res = vecDot(tmp1_inp4, tmp_ker4, 0);

        tmp0_inp4 = make_char4(inp4[5].w, inp4[6].w, inp4[8].w, inp4[9].w);
        tmp1_inp4 = make_char4(inp4[6].w, inp4[7].w, inp4[9].w, inp4[10].w);
        tmp_ker4 = make_char4(ker4[1][1].w, ker4[1][2].w, ker4[2][0].w, ker4[2][1].w);
        tmp0_res += vecDot(tmp0_inp4, tmp_ker4, 0);
        tmp1_res += vecDot(tmp1_inp4, tmp_ker4, 0);

        tmp0_res += inp4[10].w * ker4[2][2].w;
        tmp1_res += inp4[11].w * ker4[2][2].w;

        color3_0 += (float)tmp0_res;
        color3_1 += (float)tmp1_res;



        float4 scale4 = ((float4 *)(scale + oz))[0];
        color0_0 = color0_0 * scale4.x;
        color0_1 = color0_1 * scale4.x;

        color1_0 = color1_0 * scale4.y;
        color1_1 = color1_1 * scale4.y;

        color2_0 = color2_0 * scale4.z;
        color2_1 = color2_1 * scale4.z;

        color3_0 = color3_0 * scale4.w;
        color3_1 = color3_1 * scale4.w;

        color0_0 = max(color0_0, (float)minV);
        color0_0 = min(color0_0, (float)maxV);
        color0_1 = max(color0_1, (float)minV);
        color0_1 = min(color0_1, (float)maxV);

        color1_0 = max(color1_0, (float)minV);
        color1_0 = min(color1_0, (float)maxV);
        color1_1 = max(color1_1, (float)minV);
        color1_1 = min(color1_1, (float)maxV);

        color2_0 = max(color2_0, (float)minV);
        color2_0 = min(color2_0, (float)maxV);
        color2_1 = max(color2_1, (float)minV);
        color2_1 = min(color2_1, (float)maxV);

        color3_0 = max(color3_0, (float)minV);
        color3_0 = min(color3_0, (float)maxV);
        color3_1 = max(color3_1, (float)minV);
        color3_1 = min(color3_1, (float)maxV);
        int dst_offset = ((ob * oh + oy) * ow + ox) * c_p + oz;

        char4 res0 = make_char4(__float2int_rn(color0_0), __float2int_rn(color1_0), __float2int_rn(color2_0), __float2int_rn(color3_0));
        char4 res1 = make_char4(__float2int_rn(color0_1), __float2int_rn(color1_1), __float2int_rn(color2_1), __float2int_rn(color3_1));

        ((char4*)(output + dst_offset))[0] = res0;
        ((char4*)(output + dst_offset + c_p))[0] = res1;

    }
}

DepthwiseConvInt8Execution::DepthwiseConvInt8Execution(Backend* backend, const Op* op, std::shared_ptr<ConvInt8CutlassExecution::Resource> res) : ConvInt8CutlassExecution(backend, op, res) {
    mOp = op;
    mResource = res;//
}
DepthwiseConvInt8Execution::~DepthwiseConvInt8Execution() {
    // Do nothing
}

bool DepthwiseConvInt8Execution::onClone(Backend* bn, const Op* op, Execution** dst) {
    if (nullptr == dst) {
        return true;
    }
    auto exe = new DepthwiseConvInt8Execution(bn, op, mResource);
    *dst = exe;
    return true;
}

ErrorCode DepthwiseConvInt8Execution::onResize(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs) {
    auto input  = inputs[0];
    auto output = outputs[0];
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();

    std::vector<float> inputQuantInfo = TensorUtils::getQuantInfo(input);
    std::vector<float> outputQuantInfo = TensorUtils::getQuantInfo(output);
    mResource->updateInputOutputScale(inputQuantInfo, outputQuantInfo);
    runtime->memcpy(mResource->mBiasInt32Ptr, mResource->mBiasInt32Vec, mResource->mOutputChannelPack*sizeof(int32_t), MNNMemcpyHostToDevice);
    runtime->memcpy(mResource->mScaleFloatPtr, mResource->mScaleFloatVec, mResource->mOutputChannelPack*sizeof(float), MNNMemcpyHostToDevice);


    mPads = ConvolutionCommon::convolutionPad(input, output, mOp->main_as_Convolution2D()->common());

    auto mCommon = mOp->main_as_Convolution2D()->common();

    const int src_width      = input->width();
    const int src_height     = input->height();
    const int dst_width      = output->width();
    const int dst_height     = output->height();
    const int strideY        = mCommon->strideY();
    const int strideX        = mCommon->strideX();
    const int dilateY        = mCommon->dilateY();
    const int dilateX        = mCommon->dilateX();
    const int kernel_height  = mCommon->kernelY();
    const int kernel_width   = mCommon->kernelX();

    mStrides = std::make_pair(strideX, strideY);
    mDilates = std::make_pair(dilateX, dilateY);
    mKernels = std::make_pair(kernel_width, kernel_height);

    auto clamp_max = mResource->mClampMax;
    auto clamp_min = mResource->mClampMin;

    if (mCommon->relu()) {
        clamp_min = 0;
    }
    if (mCommon->relu6()) {
        clamp_min = 0;
        clamp_max = 6;
    }
    mClamps = std::make_pair(clamp_max, clamp_min);
    // MNN_PRINT("%d-%d-%d-%d, %d-%d-%d-%d\n", mKernels.first, mKernels.second, mStrides.first, mStrides.second, mDilates.first, mDilates.second, mPads.first, mPads.second);

    return NO_ERROR;
}

ErrorCode DepthwiseConvInt8Execution::onExecute(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    auto& prop = runtime->prop();

    auto input        = inputs[0];
    auto output       = outputs[0];
    const int batch   = input->batch();
    const int c       = input->channel();
    const int c_p     = UP_DIV(c, INT8_PACK_NUMBER) * INT8_PACK_NUMBER;
    const int iw      = input->width();
    const int ih      = input->height();
    const int ow      = output->width();
    const int oh      = output->height();
    const int total   = batch * c_p * oh * ow;

    const int k_p     = UP_DIV(mKernels.first * mKernels.second, INT8_PACK_NUMBER) * INT8_PACK_NUMBER;
    const auto weightPtr   = mResource->mWeightInt8Ptr;
    const auto biasPtr     = mResource->mBiasInt32Ptr;
    const auto scalePtr    = mResource->mScaleFloatPtr;

    int limitThreads = UP_DIV(total, prop.multiProcessorCount);
    int threads_num = ALIMIN(prop.maxThreadsPerBlock/2, limitThreads);
    int block_num = prop.multiProcessorCount;

    DivModFast d_oc(c_p / 4);
    DivModFast d_ow(ow);
    DivModFast d_oh(oh);


    if(mKernels.first==3 && mKernels.second==3 && mStrides.first==1 && mStrides.second==1 && mPads.first==1 && mPads.second==1 && ow % 2 ==0) {
        DivModFast d_ow2(ow/2);

        CONV_DW3x3S1_INT8_OPT<<<block_num, threads_num>>>((const int8_t*)inputs[0]->deviceId(), (const int8_t*)weightPtr,
            (const int32_t*)biasPtr, (const float*)scalePtr, (int8_t*)outputs[0]->deviceId(),
            mClamps.first, mClamps.second, iw, ih, c, c_p, ow, oh, mKernels.first, mKernels.second, k_p,
            mDilates.first, mDilates.second, mStrides.first, mStrides.second, mPads.first, mPads.second,
            total, d_oc, d_ow2, d_oh);
        checkKernelErrors;
        return NO_ERROR;
    }

    CONV_DW_INT8_<<<block_num, threads_num>>>((const int8_t*)inputs[0]->deviceId(), (const int8_t*)weightPtr,
        (const int32_t*)biasPtr, (const float*)scalePtr, (int8_t*)outputs[0]->deviceId(),
        mClamps.first, mClamps.second, iw, ih, c, c_p, ow, oh, mKernels.first, mKernels.second, k_p,
        mDilates.first, mDilates.second, mStrides.first, mStrides.second, mPads.first, mPads.second,
        total, d_oc, d_ow, d_oh);
    checkKernelErrors;

    return NO_ERROR;
}

class DepthWiseConvInt8ExecutionCreator : public CUDABackend::Creator {
public:
    virtual Execution* onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs,
                                const MNN::Op* op, Backend* backend) const override {
        if (inputs.size() > 1) {
            MNN_PRINT("OpType_DepthwiseConvInt8 CUDA not support multi input!, fall back...\n");
            return nullptr;
        }
        std::shared_ptr<ConvInt8CutlassExecution::Resource> resource(new ConvInt8CutlassExecution::Resource(backend, op));
        return new DepthwiseConvInt8Execution(backend, op, resource);
    }
};

static CUDACreatorRegister<DepthWiseConvInt8ExecutionCreator> __init(OpType_DepthwiseConvInt8);

} // namespace CUDA
} // namespace MNN
