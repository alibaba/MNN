#include "hip/hip_runtime.h"
//
//  MultiInputDeconvExecution.cpp
//  MNN
//
//  Created by MNN on 2023/04/24.
//  Copyright © 2018, Alibaba Group Holding Limited
//

#include "MultiInputDeconvExecution.hpp"
#include "ConvBaseKernel.cuh"
#include "DeconvBaseKernel.cuh"

//#define DEBUG

namespace MNN {
namespace CUDA {

MultiInputDeconvExecution::MultiInputDeconvExecution(const MNN::Op* op, Backend* backend) : CutlassDeconvCommonExecution(backend) {
    mOp = op;
    auto runtime = static_cast<CUDABackend*>(backend)->getCUDARuntime();
    mPrecisonLevel = static_cast<CUDABackend*>(backend)->getPrecision();
    mFp16Infer = (mPrecisonLevel == 2);
    mFp32Infer = (mPrecisonLevel == 1);
    mFp16Fp32MixInfer = (mPrecisonLevel == 0);
}

MultiInputDeconvExecution::~MultiInputDeconvExecution() {

}


ErrorCode MultiInputDeconvExecution::onResize(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    auto input = inputs[0], output = outputs[0];
    auto bytes = static_cast<CUDABackend*>(backend())->getBytes(inputs[0]);
    auto convCommon = mOp->main_as_Convolution2D()->common();

    // Col2Im Param
    auto pad = ConvolutionCommon::convolutionTransposePad(input, output, mOp->main_as_Convolution2D()->common());
    mCol2ImParamter.dilateX         = convCommon->dilateX();
    mCol2ImParamter.dilateY         = convCommon->dilateY();
    mCol2ImParamter.strideX         = convCommon->strideX();
    mCol2ImParamter.strideY         = convCommon->strideY();
    mCol2ImParamter.ic              = input->channel();
    mCol2ImParamter.oc              = output->channel();
    mCol2ImParamter.kernelX         = convCommon->kernelX();
    mCol2ImParamter.kernelY         = convCommon->kernelY();
    mCol2ImParamter.padX = pad.first;
    mCol2ImParamter.padY = pad.second;

    mCol2ImParamter.ih = input->height();
    mCol2ImParamter.iw = input->width();
    mCol2ImParamter.oh = output->height();
    mCol2ImParamter.ow = output->width();
    mCol2ImParamter.ob = output->batch();


    mActivationType = convCommon->relu() ? 1 : convCommon->relu6() ? 2 : 0;


    mKernelInfo.kernelX        = convCommon->kernelX();
    mKernelInfo.kernelY        = convCommon->kernelY();
    mKernelInfo.groups         = convCommon->group();
    mKernelInfo.strideX        = convCommon->strideX();
    mKernelInfo.strideY        = convCommon->strideY();
    mKernelInfo.dilateX        = convCommon->dilateX();
    mKernelInfo.dilateY        = convCommon->dilateY();
    mKernelInfo.activationType = mActivationType;
    mKernelInfo.kernelN        = output->channel();
    mKernelInfo.kernelC        = input->channel();

    // Matmul Param
    int e = output->channel() * mKernelInfo.kernelX * mKernelInfo.kernelY;
    int l = input->channel();
    int h = input->height() * input->width() * output->batch();

    mGemmInfo.elh[0] = e;
    mGemmInfo.elh[1] = l;
    mGemmInfo.elh[2] = h;
    mGemmInfo.elhPad[0] = UP_DIV(e, PACK_NUMBER) * PACK_NUMBER;
    mGemmInfo.elhPad[1] = UP_DIV(l, PACK_NUMBER) * PACK_NUMBER;
    mGemmInfo.elhPad[2] = UP_DIV(h, PACK_NUMBER) * PACK_NUMBER;


    // Alloc temp cuda memory
    auto pool = static_cast<CUDABackend*>(backend())->getBufferPool();
    std::pair<void*, size_t> buffer_input, buffer_im2col;
    if(mFp16Fp32MixInfer) {
        buffer_input = pool->alloc(sizeof(__half) * mGemmInfo.elhPad[1] * mGemmInfo.elh[2]);
        mInputBuffer = (void*)((uint8_t*)buffer_input.first + buffer_input.second);
    } else {
        mInputBuffer = (void*)input->deviceId();
    }
    buffer_im2col = pool->alloc(bytes * mGemmInfo.elh[0] * mGemmInfo.elhPad[2]);
    mIm2ColBuffer = (void*)((uint8_t*)buffer_im2col.first + buffer_im2col.second);

    mNeedWeightFill = (mGemmInfo.elh[1] != mGemmInfo.elhPad[1]);
    std::pair<void*, int> buffer_filter;
    if(mNeedWeightFill) {
        buffer_filter = pool->alloc(bytes * (size_t)mGemmInfo.elh[0] * (size_t)mGemmInfo.elhPad[1]);
        mFilterAddr = (void*)((uint8_t*)buffer_filter.first + buffer_filter.second);
    } else {
        mFilterAddr = (void*)inputs[1]->deviceId();
    }


    if(mFp16Fp32MixInfer || mFp32Infer) {
        mZeroTensor.reset(Tensor::createDevice<uint32_t>({mGemmInfo.elhPad[2]}));
    } else {
        mZeroTensor.reset(Tensor::createDevice<uint16_t>({mGemmInfo.elhPad[2]}));
    }
    static_cast<CUDABackend*>(backend())->onAcquireBuffer(mZeroTensor.get(), Backend::STATIC);

    mZeroPtr = (void *)mZeroTensor.get()->buffer().device;
    cuda_check(hipMemset(mZeroPtr, 0, mGemmInfo.elhPad[2]*bytes));


    // free for Reuse
    if(mFp16Fp32MixInfer) {
        pool->free(buffer_input);
    }
    pool->free(buffer_im2col);
    if(mNeedWeightFill) {
        pool->free(buffer_filter);
    }
 
    // Call from different function
    if(mFp32Infer){
        return callCutlassGemmCudaCoreFloat32(inputs, outputs);
    } 
 
    mGpuComputeCap = runtime->compute_capability();
    //MNN_PRINT("Gpu smArch is sm_%d\n", mGpuComputeCap);
    if(mGpuComputeCap < 75) {
        return callCutlassGemmCudaCoreFloat16(inputs, outputs);
    }
    return callCutlassGemmTensorCore(inputs, outputs);
}

ErrorCode MultiInputDeconvExecution::onExecute(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    const void *input_addr = (const void*)inputs[0]->deviceId();
    void *output_addr = (void*)outputs[0]->deviceId();
    if(inputs.size() > 2) {
        mBiasAddr = (void*)inputs[2]->deviceId();
    }
    
    // Do input convert
    if(mFp16Fp32MixInfer) {
        size_t maxCount = mGemmInfo.elhPad[1] * mGemmInfo.elh[2];
        callFloat2Half((const void*)input_addr, (void*)mInputBuffer, maxCount, runtime);
    }

    // Do weight Reoreder
    if(mNeedWeightFill) {
        callWeightReorder((const void *)inputs[1]->deviceId(), (void *)mFilterAddr, mKernelInfo, mGemmInfo.elhPad[1], mPrecisonLevel, runtime);
    }

    // Run cutlass gemm forward
    runCutlassGemmFunc();

    // Run Col2Im
    int convert_flag = mPrecisonLevel;
    if(convert_flag == 0) {
        convert_flag = 1;
    }
    callCol2ImFunc((const void*)mIm2ColBuffer, (const void*)mBiasAddr, (void *)output_addr, &mCol2ImParamter, convert_flag, runtime);
    
    return NO_ERROR;
}


}// namespace CUDA
}// namespace MNN